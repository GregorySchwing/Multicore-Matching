#include "hip/hip_runtime.h"
#include "BussKernelization.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

inline void checkLastErrorCUDA(const char *file, int line)
{
  hipError_t code = hipGetLastError();
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PrintDegrees(int nrVertices, 
                            int *ddegrees){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= nrVertices) return;
    int degree = ddegrees[threadID];
    printf("Vertex %d's deg %d\n", threadID, degree);
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PrintRowCols(int k, 
                            int *ddegrees){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= k) return;
    int degree = ddegrees[threadID];
    printf("Vertex %d's rc %d\n", threadID, degree);
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void BussKernelizationP1Kernel(int nrVertices, 
                                        int k, 
                                        int kPrime,
                                        int recursiveStackIndex, 
                                        int *ddegrees,
                                        int *dKernelSolutionRows,
                                        int *dKernelSolutionCols){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= nrVertices) return;
    int degree = ddegrees[threadID];
    if (degree <= kPrime) return;
    printf("Vertex %d's deg %d exceeds kPrime %d\n", threadID, degree, kPrime);
    int solutionIndex = atomicAdd(&dKernelSolutionRows[recursiveStackIndex], 1);
    // dsolution = new int[k];
    // Prevent oob
    if (solutionIndex >= k){
        return;
    }
    dKernelSolutionCols[solutionIndex] = threadID;
}

__global__ void ReduceDegrees(          int nrVertices,
                                        int *ddegrees,
                                        int *dremainingedges){
    extern __shared__ int temp[];
    int threadID = threadIdx.x;
    int vertexID = blockIdx.x * blockDim.x + threadIdx.x;

    // Load degrees into shared memory
    if (vertexID >= nrVertices) 
        temp[threadID] = 0;
    else
        temp[threadID] = ddegrees[vertexID];

    //printf("vertex %d degree %d\n", vertexID, temp[threadID]);
    // Warp reduce block of degrees
    
    for (int d=blockDim.x>>1; d>=1; d>>=1) {
        __syncthreads();
        if (threadID<d) temp[threadID] += temp[threadID+d];
    }
    //if (threadIdx.x == 0) printf("SumOfEdges %d\n", temp[threadID]);
    // Add block reduced value to global value.
    if (threadIdx.x == 0) atomicAdd(&dremainingedges[0], temp[threadID]);
    

}

__global__ void BussKernelizationP2Kernel(int sizeOfKernelSolution,
                                        int *ddegrees,
                                        int *dremainingedges,
                                        int *dkernelsolution){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= sizeOfKernelSolution) return;
    int solnVertex = dkernelsolution[threadID];
    int degree = ddegrees[solnVertex];
    int remainingedges = atomicSub(&dremainingedges[0], degree);
    //printf("Removed %d's %d edges : edges remaining %d\n", solnVertex, degree/2, remainingedges/2);
}

/*
BussKernelization::BussKernelization(const mtc::Graph &_graph, 
                                    const int &_threadsPerBlock, 
                                    const unsigned int &_barrier, 
                                    const unsigned int &_k,
                                    bool &_solutionCantExist):
    graph(_graph),
    threadsPerBlock(_threadsPerBlock),
    barrier(_barrier),
    k(_k),
    solutionCantExist(_solutionCantExist){


    if (hipMalloc(&dkernelsolution, sizeof(int)*k) != hipSuccess || 
        hipMalloc(&ddegrees, sizeof(int)*graph.nrVertices) != hipSuccess || 
        hipMalloc(&dremainingedges, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dsizeofkernelsolution, sizeof(int)*1) != hipSuccess)	
    {
		std::cerr << "Not enough memory on device!" << std::endl;
		throw std::exception();
	}

    sizeOfKernelSolution = 0;
    bussKernelizationP1();
    if (sizeOfKernelSolution > k){
        printf("|S| = b (%d) > k (%d), no solution exists\n", sizeOfKernelSolution, k);
    } else {
        printf("|S| = b (%d) <= k (%d), a solution may exist\n", sizeOfKernelSolution, k);
    }
    solutionCantExist = sizeOfKernelSolution > k;
    kPrime = k - sizeOfKernelSolution;
    if(!solutionCantExist){
        printf("Setting k' = k %d - b %d = %d\n", k, sizeOfKernelSolution, kPrime);
        bussKernelizationP2();
        solutionCantExist = remainingedges > k*kPrime;
        if(remainingedges > k*kPrime){
            printf("|G'(E)| (%d) > k (%d) * k' (%d) = %d, no solution exists\n",remainingedges, k, kPrime, k*kPrime);
        } else {
            printf("|G'(E)| (%d) <= k (%d) * k' (%d) = %d, a solution may exist\n",remainingedges, k, kPrime, k*kPrime);
        }
    }
}
*/

void BussKernelization::PerformBussKernelization(int nrVertices,
                                                int threadsPerBlock,
                                                const int k,
                                                int & kPrime,
                                                int recursiveStackIndex,
                                                int * ddegrees,
                                                int * dkernelsolutionrows,
                                                int * dkernelsolutioncols,
                                                int * deviceRemainingEdges,
                                                bool & solutionCantExist){
	// Should calculate an current degree if interleaving.
	// If not interleaving this can be commented out.
	//hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(ddegrees),  0, size_t(graph.nrVertices));
	// If interleaving, call a CalcDegreesKernel here.

    int sizeOfKernelSolution = 0;
    // Copy last number of kernel soln vertices 
    int lastTwoEntriesInKSR[2];
    bussKernelizationP1(nrVertices,
                        threadsPerBlock, 
                        k,
                        kPrime,
                        sizeOfKernelSolution,
                        recursiveStackIndex,
                        ddegrees,
                        dkernelsolutionrows,
                        dkernelsolutioncols);
    hipMemcpy(lastTwoEntriesInKSR, &dkernelsolutionrows[recursiveStackIndex-1], sizeof(int)*2, hipMemcpyDeviceToHost);
    sizeOfKernelSolution = lastTwoEntriesInKSR[1] - lastTwoEntriesInKSR[0];
    if (sizeOfKernelSolution > k){
        printf("|S| = b (%d) > k (%d), no solution exists\n", sizeOfKernelSolution, k);
    } else {
        printf("|S| = b (%d) <= k (%d), a solution may exist\n", sizeOfKernelSolution, k);
    }
    solutionCantExist = sizeOfKernelSolution > k;
    kPrime -= sizeOfKernelSolution;
    if(!solutionCantExist){
        printf("Setting k' = k %d - b %d = %d\n", k, sizeOfKernelSolution, kPrime);
        int remainingedges = 0;
        bussKernelizationP2(nrVertices,
                            threadsPerBlock, 
                            recursiveStackIndex,
                            remainingedges,
                            sizeOfKernelSolution,
                            ddegrees,
                            deviceRemainingEdges,
                            lastTwoEntriesInKSR[0],
                            dkernelsolutioncols);
        solutionCantExist = remainingedges > k*kPrime;
        if(remainingedges > k*kPrime){
            printf("|G'(E)| (%d) > k (%d) * k' (%d) = %d, no solution exists\n",remainingedges, k, kPrime, k*kPrime);
        } else {
            printf("|G'(E)| (%d) <= k (%d) * k' (%d) = %d, a solution may exist\n",remainingedges, k, kPrime, k*kPrime);
        }
    }
}

// Initial kernelization before search tree is built
void BussKernelization::bussKernelizationP1(int nrVertices,
                                            int threadsPerBlock,
                                            int k,
                                            int kPrime,
                                            int & sizeOfKernelSolution,
                                            int recursiveStackIndex,
                                            int * dDegrees,
                                            int * dKernelSolutionRows,
                                            int * dKernelSolutionCols){

    //hipMemcpy(&dKernelSolutionRows[recursiveStackIndex], &dKernelSolutionRows[recursiveStackIndex-1], sizeof(int)*1, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);	
    int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    BussKernelizationP1Kernel<<<blocksPerGrid, threadsPerBlock>>>(nrVertices, 
                                                                k, 
                                                                kPrime,
                                                                recursiveStackIndex,
                                                                dDegrees,
                                                                dKernelSolutionRows,
                                                                dKernelSolutionCols);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);	
}

// Initial kernelization before search tree is built
void BussKernelization::bussKernelizationP2(int nrVertices,
                                        int threadsPerBlock,
                                        int recursiveStackIndex,
                                        int & remainingEdges,
                                        int sizeOfKernelSolution,
                                        int * dDegrees,
                                        int * deviceRemainingEdges,
                                        int startOfNewKernel,
                                        int * dkernelsolutioncols)
{
    int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);	
    ReduceDegrees<<<blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(nrVertices,
                                                                                    dDegrees,
                                                                                    deviceRemainingEdges);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);	
    hipMemcpy(&remainingEdges, deviceRemainingEdges, sizeof(int)*1, hipMemcpyDeviceToHost);
    printf("Remaining edges before Kernel %d\n", remainingEdges/2);
    // Using the indices to calculate degrees requires doubling and then halving
    // Since each edge is counted twice, once in each connecting vertex's indices.x to indices.y
    //remainingedges = 2*graph.nrEdges;
    //hipMemcpy(dremainingedges, &remainingedges, sizeof(int)*1, hipMemcpyHostToDevice);
    blocksPerGrid = (sizeOfKernelSolution + threadsPerBlock - 1)/threadsPerBlock;
    //printf("Launching %d blocks for a solution of size %d\n", blocksPerGrid, sizeOfKernelSolution);
    BussKernelizationP2Kernel<<<blocksPerGrid, threadsPerBlock>>>(sizeOfKernelSolution,
                                                                dDegrees,
                                                                deviceRemainingEdges,
                                                                &dkernelsolutioncols[startOfNewKernel]);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);	
    hipMemcpy(&remainingEdges, deviceRemainingEdges, sizeof(int)*1, hipMemcpyDeviceToHost);
    // Using the indices to calculate degrees requires doubling and then halving
    // Since each edge is counted twice, once in each connecting vertex's indices.x to indices.y
    remainingEdges/=2;
    printf("Remaining edges after Kernel %d\n", remainingEdges);
    
}

// Initial kernelization before search tree is built
/*
int BussKernelization::GetKPrime(){
    return kPrime;
}

// Initial kernelization before search tree is built
int* BussKernelization::GetKernelSolution(){
    return dkernelsolution;
}
*/