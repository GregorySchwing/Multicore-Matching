#include "hip/hip_runtime.h"
/*
Copyright 2011, Bas Fagginger Auer.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "vcgpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "hipcub/hipcub.hpp"

using namespace std;
using namespace mtc;



#include <iostream>


// RE?
#include <curses.h>
#include <ncurses.h>

inline void checkLastErrorCUDA(const char *file, int line)
{
  hipError_t code = hipGetLastError();
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

//==== Kernel variables ====
__device__ int dkeepMatching;

texture<int2, hipTextureType1D, hipReadModeElementType> neighbourRangesTexture;
texture<int, hipTextureType1D, hipReadModeElementType> neighboursTexture;
texture<float, hipTextureType1D, hipReadModeElementType> weightsTexture;

VCGPU::VCGPU(const Graph &_graph, const int &_threadsPerBlock, const unsigned int &_barrier, const unsigned int &_k) :
		graph(_graph),
        threadsPerBlock(_threadsPerBlock),
        barrier(_barrier),
		matcher(_graph, _threadsPerBlock, _barrier),
        dfll(_graph.nrVertices),
        dbll(_graph.nrVertices),
        k(_k),
        depthOfSearchTree(_k/2)
{
    finishedLeavesPerLevel.resize(depthOfSearchTree+1);
    totalLeavesPerLevel.resize(depthOfSearchTree+1);

    sizeOfSearchTree = CalculateSpaceForDesiredNumberOfLevels(depthOfSearchTree);
    printf("SIZE OF SEARCH TREE %lld\n", sizeOfSearchTree);
    searchtree.resize(sizeOfSearchTree);

    // Wrong since numEdges < neighbors (up to double the num edges, in and out)
    //hipMalloc(&dedgestatus, sizeof(int)*graph.nrEdges) != hipSuccess || 
    if (hipMalloc(&dedgestatus, sizeof(int)*graph.neighbours.size()) != hipSuccess || 
        hipMalloc(&dlength, sizeof(int)*graph.nrVertices) != hipSuccess || 
        hipMalloc(&dsearchtree, sizeof(int2)*sizeOfSearchTree) != hipSuccess || 
        hipMalloc(&dfullpathcount, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dnumleaves, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&ddynamicallyaddedvertices, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dfinishedLeavesPerLevel, sizeof(float)*depthOfSearchTree) != hipSuccess || 
        //hipMalloc(&active_frontier_status, sizeof(int)*depthOfSearchTree) != hipSuccess || 
        hipMalloc(&ddegrees, sizeof(int)*graph.nrVertices) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}
    edgestatus.resize(graph.neighbours.size());
    newdegrees.resize(graph.nrVertices);
    ReinitializeArrays();
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)matcher.dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)matcher.dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());


}

VCGPU::~VCGPU(){
    hipFree(ddegrees);
	hipFree(dlength);
    hipFree(dsearchtree);
    hipFree(dedgestatus);
    hipFree(dfullpathcount);
    hipFree(dnumleaves);
    hipFree(ddynamicallyaddedvertices);
	hipUnbindTexture(neighboursTexture);
	hipUnbindTexture(neighbourRangesTexture);
}

long long VCGPU::CalculateSpaceForDesiredNumberOfLevels(int NumberOfLevels){
    long long summand= 0;
    // ceiling(vertexCount/2) loops
    for (int i = 0; i <= NumberOfLevels; ++i){
        summand += pow (3.0, i);
        finishedLeavesPerLevel[i] = 0;
        totalLeavesPerLevel[i] = pow (3.0, i);
    }
    return summand;
}

void VCGPU::GetDeviceVectors(int nrVertices, std::vector<int> & fll, std::vector<int> & bll, std::vector<int> & length)
{
	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&fll[0], dforwardlinkedlist, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess ||
		hipMemcpy(&bll[0], dbackwardlinkedlist, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(&length[0], dlength, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}
}
void VCGPU::GetLengthStatistics(int nrVertices, int threadsPerBlock, int *dbackwardlinkedlist, int *dlength, int *dreducedlength)
{
	int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
	ReducePathLengths<<<blocksPerGrid, threadsPerBlock>>>(nrVertices, dbackwardlinkedlist, dlength, dreducedlength);
}

int4 VCGPU::numberCompletedPaths(int nrVertices, 
                        int leafIndex,
                        int *dbackwardlinkedlist, 
                        int *dlength){
	int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    PopulateSearchTree<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            sizeOfSearchTree, 
                                                            leafIndex,
                                                            dfinishedLeavesPerLevel,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist, 
                                                            dlength,
                                                            dfullpathcount,
                                                            dsearchtree);

    DetectAndSetPendantPathsCase3<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                        matcher.dmatch,
                                                        dfinishedLeavesPerLevel,
                                                        dforwardlinkedlist,
                                                        dbackwardlinkedlist,
                                                        dedgestatus, 
                                                        dlength,
                                                        ddynamicallyaddedvertices);
    DetectAndSetPendantPathsCase4<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                        matcher.dmatch,
                                                        dfinishedLeavesPerLevel,
                                                        dforwardlinkedlist,
                                                        dbackwardlinkedlist,
                                                        dedgestatus, 
                                                        dlength,
                                                        ddynamicallyaddedvertices);

    hipMemcpy(&fullpathcount, &dfullpathcount[0], sizeof(int)*1, hipMemcpyDeviceToHost);
    
    int4 myActiveLeaves = CalculateLeafOffsets(leafIndex,
                                                fullpathcount);
    //printf("My active leaves %d %d %d %d\n", myActiveLeaves.x, myActiveLeaves.y, myActiveLeaves.z, myActiveLeaves.w);
    return myActiveLeaves;
}

// 2 Possibilities for recycling the paths of length 1&2
// Depending on whether we want to perform parallel frontier splitting.

// PFS (NO) - this is my first implementation.
// 1) Continue matching from a leaf, after removing edges 
// of included vertices and un-coloring the non-included vertices
// This approach will prioritize DF growth of the search tree.
// This allows us to only maintain 1 copy of the edge status in GPU mem.

// PFS (YES) - can try this in second implementation.
// 2) Evaluate each path for pendantness at each leaf node.
// This requires storing the edge status of each leaf node
// for any benefit to be seen.

//  However, it is MUCH easier to just wipe away intermediate paths
// And perform BFS at each leaf node, if we stick to BFS for as 
// long as complete levels can be formed.

// Will most likely copy back the frontier bool array and iterate through each frontier one at a time
// for v1.
//for (int activeRoot = leftMostLeafOfLevel; activeRoot < rightMostLeafOfLevel; ++activeRoot){

// For DFS, we'd assume that every level beneath the last BFS has
// started at its left most child, and will need to be recursively
// searched from the bottom.

void VCGPU::FindCover(int root){
    // If you want to check the quality of each match, uncomment
    // Else, the only noticable changes will be in the recursion stack 
    // and the device search tree.
    //std::vector<int> match;
    //matcher.initialMatching(match);
    int depthOfLeaf = ceil(logf(2*root + 1) / logf(3)) - (int)(root==0);
    if (depthOfLeaf >= depthOfSearchTree)
        return;
//    printf("\033[A\33[2K\rCalling Find Cover from %d, level depth of leaf %d\n", root, depthOfLeaf);
    hipMemcpy(&finishedLeavesPerLevel[0], dfinishedLeavesPerLevel, sizeof(float)*depthOfSearchTree, hipMemcpyDeviceToHost);
/*
    curs_set (0);
    for(int i = 0; i < depthOfSearchTree; ++i){
        mvprintw (i, 4, "Depth %d %f Complete\n", i, finishedLeavesPerLevel[i]/totalLeavesPerLevel[i]);
    }
    refresh ();
*/
    ReinitializeArrays();
    SetEdgesOfLeaf(root);
    Match();
    //matcher.copyMatchingBackToHost(match);
    // Need to pass device pointer to LOP
    int4 newLeaves = numberCompletedPaths(graph.nrVertices, root, dbackwardlinkedlist, dlength); 
    hipMemcpy(&edgestatus[0], dedgestatus, sizeof(int)*graph.neighbours.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&newdegrees[0], ddegrees, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
    #ifndef NDEBUG
    PrintData (); 
    #endif
    //char temp;
    //cin >> temp;
    while(newLeaves.x < newLeaves.y){
        FindCover(newLeaves.x);
        ++newLeaves.x;
    }
    while(newLeaves.z < newLeaves.w){
        FindCover(newLeaves.z);
        ++newLeaves.z;
    }

    if (root == 0){
        hipMemcpy(&searchtree[0], dsearchtree, sizeof(int2)*searchtree.size(), hipMemcpyDeviceToHost);
  
    }

}


void VCGPU::SetEdgesOfLeaf(int leafIndex){
    // Root of search tree is empty.
    if (leafIndex == 0)
        return;
	int blocksPerGrid = 2*(ceil(logf(2*leafIndex + 1) / logf(3)) - (int)(leafIndex==0));
    SetEdges<<<blocksPerGrid, threadsPerBlock>>>(leafIndex,
                                                dedgestatus,
                                                ddegrees,
                                                dsearchtree);
	blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    CalculateDegrees<<<blocksPerGrid, threadsPerBlock>>>(graph.nrVertices,
                                                dedgestatus,
                                                ddegrees);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);
}

void VCGPU::PrintData (){
   
    printf("neighbours size %d num edges %d\n",  graph.neighbours.size(), graph.nrEdges);
    printf("Row Offs\n");
    for (int i = 0; i < graph.nrVertices; ++i){
        printf("%d %d, ",graph.neighbourRanges[i].x, graph.neighbourRanges[i].y);
    }
    printf("\n");
    printf("Cols\n");
    for (int i = 0; i < graph.neighbours.size(); ++i){
        printf("%d ",graph.neighbours[i]);
    }
    printf("\n");
    printf("Vals\n");
    for (int i = 0; i < graph.neighbours.size(); ++i){
        printf("%d ",edgestatus[i]);
    }
    printf("\n");
    printf("Degrees\n");
    for (int i = 0; i < graph.nrVertices+1; ++i){
        printf("%d ", newdegrees[i]);
    }
}
void VCGPU::Match(){
    //Initialise timers.
    hipEvent_t t0, t1, t2, t3;
    float time0, time1;

    hipEventCreate(&t0);
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    
    hipEventRecord(t0, 0);
    hipEventSynchronize(t0);

    matcher.performMatching(&matcher.match[0], t1, t2, dforwardlinkedlist, dbackwardlinkedlist, dlength, ddegrees, dedgestatus);
    
    hipEventElapsedTime(&time1, t1, t2);
    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    //Measure the total elapsed time (including data transfer) and the calculation time.
    hipEventElapsedTime(&time0, t0, t3);
    hipEventElapsedTime(&time1, t1, t2);
    //Destroy timers.
    hipEventDestroy(t3);
    hipEventDestroy(t2);
    hipEventDestroy(t1);
    hipEventDestroy(t0);
}

void VCGPU::ReinitializeArrays(){
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dedgestatus),  1, size_t(graph.neighbours.size()));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dlength),  0, size_t(graph.nrVertices));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dfullpathcount),  0, size_t(1));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dnumleaves),  0, size_t(1));
    // Only >= 0 are heads of full paths
    // Before implementing recursive backtracking, I can keep performing this memcpy to set degrees
    // and the remove tentative vertices to check a cover.
    hipMemcpy(ddegrees, &graph.degrees[0], sizeof(int)*graph.nrVertices, hipMemcpyHostToDevice);

	thrust::sequence(dfll.begin(),dfll.end());
	dforwardlinkedlist = thrust::raw_pointer_cast(&dfll[0]);
	
	thrust::sequence(dbll.begin(),dbll.end());
	dbackwardlinkedlist = thrust::raw_pointer_cast(&dbll[0]);
}


// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PopulateSearchTree(int nrVertices, 
                                    int depthOfSearchTree,
                                                int leafIndex,
                                                float * dfinishedLeavesPerLevel,
                                                int *dforwardlinkedlist, 
                                                int *dbackwardlinkedlist, 
                                                int *dlength, 
                                                int *dfullpathcount,
                                                int2* dsearchtree){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 3 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];
    int fourth = dforwardlinkedlist[third];

    int leavesToProcess = atomicAdd(&dfullpathcount[0], 1) + 1;
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // Solved for leavesToProcess < closed form
    // start from level 1, hence add a level if LTP > 0, 1 complete level 
    // Add 1 if LTP == 0 to prevent runtime error
    // LTP = 2
    // CL = 1
    // Always add 2 to prevent run time error, also to start counting at level 1 not level 0
    int incompleteLevel = ceil(logf(2*leavesToProcess + 1) / logf(3)) - (int)(leavesToProcess==0);
    int arbitraryParameter = 3*((3*leafIndex)+1);
    int leftMostLeafIndexOfIncompleteLevel = ((2*arbitraryParameter+3)*powf(3.0, incompleteLevel-1) - 3)/6;

    int leavesFromIncompleteLevelLvl = powf(3.0, incompleteLevel); 
    int treeSizeNotIncludingThisLevel = (1.0 - powf(3.0, (incompleteLevel-1)))/(1.0 - 3.0);  
    // Test from root for now, this code can have an arbitrary root though
    //leafIndex = global_active_leaves[globalIndex];
//    leafIndex = 0;
    // Closed form solution of recurrence relation shown in comment above method
    // Subtract 1 because reasons
    int internalLeafIndex = leavesToProcess - 1 - treeSizeNotIncludingThisLevel;
    int levelOffset = leftMostLeafIndexOfIncompleteLevel + 3*internalLeafIndex;
    #ifndef NDEBUG
    printf("Level Depth %d\n", incompleteLevel);
    printf("Level Width  %d\n", leavesFromIncompleteLevelLvl);
    printf("Size of Tree %d\n", treeSizeNotIncludingThisLevel);
    printf("Global level left offset (GLLO) %d\n", leftMostLeafIndexOfIncompleteLevel);
    printf("internalLeafIndex %d\n", internalLeafIndex);
    printf("Displacement frok GLLO %d %d %d \n", levelOffset,
                                                levelOffset + 1,
                                                levelOffset + 2);
    #endif
    int depthOfLeaf = ceil(logf(2*levelOffset + 2 + 1) / logf(3)) - (levelOffset == 0);
    if (depthOfLeaf > depthOfSearchTree){
        return;
    }
    // Test from root for now, this code can have an arbitrary root though
    dsearchtree[levelOffset + 0] = make_int2(first, third);
    dsearchtree[levelOffset + 1] = make_int2(second, third);
    dsearchtree[levelOffset + 2] = make_int2(second, fourth);
    // Add to device pointer of level
    atomicAdd(&dfinishedLeavesPerLevel[depthOfLeaf], 3); 
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void DetectAndSetPendantPathsCase4(int nrVertices, 
                                                int *match, 
                                                int *dforwardlinkedlist, 
                                                int *dbackwardlinkedlist, 
                                                int * dedgestatus,
                                                int *dlength, 
                                                int *dnumberofpendantvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 1 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];

    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 2){
        SetEdges(first, dedgestatus);
    } else if (match[second] == 2){
        SetEdges(second, dedgestatus);
    }
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void DetectAndSetPendantPathsCase3(int nrVertices, 
                                                int *match, 
                                                int *dforwardlinkedlist, 
                                                int *dbackwardlinkedlist, 
                                                int * dedgestatus,
                                                int *dlength, 
                                                int *ddynamicallyaddedvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 2 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];

    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 2){
        SetEdges(first, dedgestatus);
    } else if (match[third] == 2){
        SetEdges(third, dedgestatus);
    }
}

// Makes sense for BFS
// For DFS use Recursive Backtracking
__global__ void GetFrontierStatus(int nrNodes,
							int *active_frontier_status){

}

__global__ void ReducePathLengths(int nrVertices,
							int *dbackwardlinkedlist,
                            int* dlength,
                            int* dreducedlength){}

__global__ void SetEdges(const int leafIndex,
                        int * dedgestatus,
                        int * ddegrees,
                        int2 *dsearchtree){

	//Determine blue and red groups using MD5 hashing.
	//Based on the Wikipedia MD5 hashing pseudocode (http://en.wikipedia.org/wiki/MD5).
	const int numberOfLevelsToAscend = blockIdx.x/2;
    //if (threadIdx.x == 0){
    int thisBlocksSearchTreeNode = leafIndex / pow (3.0, numberOfLevelsToAscend);
    //}
    int2 verticesInNode = dsearchtree[thisBlocksSearchTreeNode];
    int i;
    if (blockIdx.x % 2 == 0)
        i = verticesInNode.x;
    else 
        i = verticesInNode.y;
    int2 indices = tex1Dfetch(neighbourRangesTexture, i);
    #ifndef NDEBUG
    if (threadIdx.x == 0){
        printf("thisBlocksSearchTreeNode %d\n", thisBlocksSearchTreeNode);
        printf("Setting vertex %d\n", i);
        printf("Turning off edges between %d and %d in col array\n",indices.x,indices.y);
    }
    #endif
    for (int j = indices.x + threadIdx.x; j < indices.y; j += blockDim.x){
        //const int ni = tex1Dfetch(neighboursTexture, j);
        //printf("Turning off edge %d which is index %d of the val array\n",ni,j);
        // Set out-edges
        dedgestatus[j] = 0;
    }   
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    bool foundChild, tmp;
    // There are two possibilities for parallelization here:
    // 1) Each thread will take an out edge, and then each thread will scan the edges leaving 
    // that vertex for the original vertex.
    //for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){

    // Basically, each thread is reading wildly different data
    // 2) 1 out edge is traversed at a time, and then all the threads scan
    // all the edges leaving that vertex for the original vertex.
    // This is the more favorable data access pattern.
    const int2 indices_curr = tex1Dfetch(neighbourRangesTexture, i);
    for (int j = indices_curr.x + threadIdx.x; j < indices_curr.y; j += blockDim.x){
        const int ni = tex1Dfetch(neighboursTexture, j);    
        const int2 indices_neighbor = tex1Dfetch(neighbourRangesTexture, ni);
          for (int j_n = indices_neighbor.x; j_n < indices_neighbor.y; ++j_n){
                const int nj = tex1Dfetch(neighboursTexture, j_n);       
                foundChild = i == nj;
                // Set in-edge
                // store edge status
                tmp = dedgestatus[j_n];
                //   foundChild     tmp   (foundChild & tmp)  (foundChild & tmp)^tmp
                //1)      0          0            0                       0
                //2)      1          0            0                       0
                //3)      0          1            0                       1
                //4)      1          1            1                       0
                //
                // Case 1: isnt myChild and edge is off, stay off
                // Case 2: is myChild and edge is off, stay off
                // Case 3: isn't myChild and edge is on, stay on
                // Case 4: is myChild and edge is on, turn off
                // All this logic is necessary because we aren't using degree to set upperbound
                // we are using row offsets, which may include some edges turned off on a previous
                // pendant edge processing step.
                // Doesnt work for some reason
                // dedgestatus[j_n] ^= (foundChild & tmp);

                if(foundChild && tmp)
                    dedgestatus[j] = 0;
        }
    } 
}

__device__ void SetEdges(   int vertexToInclude,
                            int * dedgestatus){

    int2 indices = tex1Dfetch(neighbourRangesTexture, vertexToInclude);
    for (int j = indices.x; j < indices.y; j += 1){
        //const int ni = tex1Dfetch(neighboursTexture, j);
        //printf("Turning off edge %d which is index %d of the val array\n",ni,j);
        // Set out-edges
        dedgestatus[j] = 0;
    }   
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    bool foundChild, tmp;
    // There are two possibilities for parallelization here:
    // 1) Each thread will take an out edge, and then each thread will scan the edges leaving 
    // that vertex for the original vertex.
    //for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){

    // Basically, each thread is reading wildly different data
    // 2) 1 out edge is traversed at a time, and then all the threads scan
    // all the edges leaving that vertex for the original vertex.
    // This is the more favorable data access pattern.
    const int2 indices_curr = tex1Dfetch(neighbourRangesTexture, vertexToInclude);
    for (int j = indices_curr.x; j < indices_curr.y; j += 1){
        const int ni = tex1Dfetch(neighboursTexture, j);    
        const int2 indices_neighbor = tex1Dfetch(neighbourRangesTexture, ni);
          for (int j_n = indices_neighbor.x; j_n < indices_neighbor.y; ++j_n){
                const int nj = tex1Dfetch(neighboursTexture, j_n);       
                foundChild = vertexToInclude == nj;
                // Set in-edge
                // store edge status
                tmp = dedgestatus[j_n];
                //   foundChild     tmp   (foundChild & tmp)  (foundChild & tmp)^tmp
                //1)      0          0            0                       0
                //2)      1          0            0                       0
                //3)      0          1            0                       1
                //4)      1          1            1                       0
                //
                // Case 1: isnt myChild and edge is off, stay off
                // Case 2: is myChild and edge is off, stay off
                // Case 3: isn't myChild and edge is on, stay on
                // Case 4: is myChild and edge is on, turn off
                // All this logic is necessary because we aren't using degree to set upperbound
                // we are using row offsets, which may include some edges turned off on a previous
                // pendant edge processing step.
                // Doesnt work for some reason
                // dedgestatus[j_n] ^= (foundChild & tmp);

                if(foundChild && tmp)
                    dedgestatus[j] = 0;
        }
    } 
}


__global__ void CalculateDegrees(
                        int nrVertices,
                        int * dedgestatus,
                        int * ddegrees){

	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices ) return;
    int sum = 0;
    int2 indices = tex1Dfetch(neighbourRangesTexture, threadID);
    for (int j = indices.x; j < indices.y; ++j){
        sum += dedgestatus[j];
    }
    ddegrees[threadID] = sum;
}

__global__ void InitDegrees(const int nrVertices,
                            int * ddegrees){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= nrVertices) return;
    const int2 indices = tex1Dfetch(neighbourRangesTexture, i);
    ddegrees[i] = indices.y - indices.x;
}

__global__ void CalculateNumberOfLeaves(int *dfullpathcount){

}


//int leafIndex = global_active_leaf_value[leafIndex];
// Solve recurrence relation 
// g(n) = 1/6*((2*C+3)*3^n - 3)
// C depends on leafIndex
// where g(0) = left-most child of depth 1
// where g(1) = left-most child of depth 2
// where g(2) = left-most child of depth 3
// ...
//int arbitraryParameter = 3*(3*leafIndex)+1);

// currently a single root is expanded in gpu memory at a time. 
// efforts were made in the FPT-kVC "done" branch to maintain multiple copies of the graph
// and explore the search tree in parallel.
int4 CalculateLeafOffsets(              int leafIndex,
                                        int fullpathcount){
    int arbitraryParameter;
    int leftMostLeafIndexOfFullLevel;
    int leftMostLeafIndexOfIncompleteLevel;
    int leavesToProcess = fullpathcount;
    if (leavesToProcess == 0)
        return make_int4( leafIndex,
                          leafIndex,
                          leafIndex,
                          leafIndex);
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // Solved for leavesToProcess < closed form
    // start from level 1, hence add a level if LTP > 0, 1 complete level 
    // Add 1 if LTP == 0 to prevent runtime error
    // LTP = 2
    // CL = 1
    // Always add 2 to prevent run time error, also to start counting at level 1 not level 0
    int completeLevel = floor(logf(2*leavesToProcess + 1) / logf(3)) - (int)(leavesToProcess==0);
    // If LTP == 0, we dont want to create any new leaves
    // Therefore, we dont want to enter the for loops.
    // The active leaf writes itself as it's parent before the for loops
    // This is overwritten within the for loops if LTP > 0
    // CLL = 3
    int leavesFromCompleteLvl = powf(3.0, completeLevel) - (int)(leavesToProcess == 0);
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // Solved for closed form < leavesToProcess
    // Always add 2 to prevent run time error, also to start counting at level 1 not level 0
    // IL = 1
    int incompleteLevel = ceil(logf(2*leavesToProcess + 1) / logf(3)) - (int)(leavesToProcess==0);
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // Add 1 when leavesToProcess isn't 0, so we start counting from level 1
    // Also subtract the root, so we start counting from level 1
    // TSC = 3
    int treeSizeComplete = (1.0 - powf(3.0, completeLevel+(int)(leavesToProcess != 0)))/(1.0 - 3.0) - (int)(leavesToProcess != 0);
    // How many internal leaves to skip in complete level
    // RFC = 1
    int removeFromComplete = ((3*leavesToProcess - treeSizeComplete) + 3 - 1) / 3;
    // Leaves that are used in next level
    int leavesFromIncompleteLvl = 3*removeFromComplete;
    
    // Test from root for now, this code can have an arbitrary root though
    arbitraryParameter = 3*((3*leafIndex)+1);
    // Closed form solution of recurrence relation shown in comment above method
    // Subtract 1 because reasons
    leftMostLeafIndexOfFullLevel = ((2*arbitraryParameter+3)*powf(3.0, completeLevel-1) - 3)/6;
    leftMostLeafIndexOfIncompleteLevel = ((2*arbitraryParameter+3)*powf(3.0, incompleteLevel-1) - 3)/6;

    int totalNewActive = (leavesFromCompleteLvl - removeFromComplete) + leavesFromIncompleteLvl;
    #ifndef NDEBUG
    printf("Leaves %d, completeLevel Level Depth %d\n",leavesToProcess, completeLevel);
    printf("Leaves %d, incompleteLevel Level Depth %d\n",leavesToProcess, incompleteLevel);
    printf("Leaves %d, treeSizeComplete %d\n",leavesToProcess, treeSizeComplete);
    printf("Leaves %d, totalNewActive %d\n",leavesToProcess, totalNewActive);
    printf("Leaves %d, leavesFromCompleteLvl %d\n",leavesToProcess, leavesFromCompleteLvl);
    printf("Leaves %d, leavesFromIncompleteLvl %d\n",leavesToProcess, leavesFromIncompleteLvl);
    printf("Leaves %d, leftMostLeafIndexOfFullLevel %d\n",leavesToProcess, leftMostLeafIndexOfFullLevel);
    printf("Leaves %d, leftMostLeafIndexOfIncompleteLevel %d\n",leavesToProcess, leftMostLeafIndexOfIncompleteLevel);
    #endif
    // Grow tree leftmost first, so put the incomplete level first.
    // Shape of leaves
    //CL    -     -    o o o 
    //IL  o o o o o o
    return make_int4( leftMostLeafIndexOfIncompleteLevel,
                        leftMostLeafIndexOfIncompleteLevel + leavesFromIncompleteLvl,
                        leftMostLeafIndexOfFullLevel,
                        leftMostLeafIndexOfFullLevel + leavesFromCompleteLvl);

}