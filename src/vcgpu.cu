#include "hip/hip_runtime.h"
/*
Copyright 2011, Bas Fagginger Auer.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "vcgpu.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "hipcub/hipcub.hpp"

using namespace std;
using namespace mtc;

#include <iostream>


// RE?
#include <curses.h>
#include <ncurses.h>

inline void checkLastErrorCUDA(const char *file, int line)
{
  hipError_t code = hipGetLastError();
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

//==== Kernel variables ====
__device__ int dkeepMatching;

texture<int2, hipTextureType1D, hipReadModeElementType> neighbourRangesTexture;
texture<int, hipTextureType1D, hipReadModeElementType> neighboursTexture;
texture<float, hipTextureType1D, hipReadModeElementType> weightsTexture;

VCGPU::VCGPU(const Graph &_graph, 
             const int &_threadsPerBlock, 
             const unsigned int &_barrier, 
             const unsigned int &_k,
             bool & _solutionCantExist) :
		graph(_graph),
        threadsPerBlock(_threadsPerBlock),
        barrier(_barrier),
		matcher(_graph, _threadsPerBlock, _barrier),
        dfll(_graph.nrVertices),
        dbll(_graph.nrVertices),
        k(_k),
        solutionCantExist(_solutionCantExist)
{
    solution.resize(k);
    if (hipMalloc(&ddegrees, sizeof(int)*graph.nrVertices) != hipSuccess || 
        hipMalloc(&dsolution, sizeof(int)*k) != hipSuccess || 
        hipMalloc(&dremainingedges, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dsizeofkernelsolution, sizeof(int)*1))	
    {
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}
    bussKernelizationP1();
    if (sizeOfKernelSolution > k){
        printf("|S| = b (%d) > k (%d), no solution exists\n", sizeOfKernelSolution, k);
    } else {
        printf("|S| = b (%d) <= k (%d), a solution may exist\n", sizeOfKernelSolution, k);
    }
    solutionCantExist = sizeOfKernelSolution > k;
    kPrime = k - sizeOfKernelSolution;
    if(!solutionCantExist){
        printf("Setting k' = k %d - b %d = %d\n", k, sizeOfKernelSolution, kPrime);
        bussKernelizationP2();
        solutionCantExist = remainingedges > k*kPrime;
        if(remainingedges > k*kPrime){
            printf("|G'(E)| (%d) > k (%d) * k' (%d) = %d, no solution exists\n",remainingedges, k, kPrime, k*kPrime);
        } else {
            printf("|G'(E)| (%d) <= k (%d) * k' (%d) = %d, a solution may exist\n",remainingedges, k, kPrime, k*kPrime);
        }
    }
    depthOfSearchTree = kPrime/2;
    if (solutionCantExist){
        return;
    }
    finishedLeavesPerLevel.resize(depthOfSearchTree+1);
    totalLeavesPerLevel.resize(depthOfSearchTree+1);
    sizeOfSearchTree = CalculateSpaceForDesiredNumberOfLevels(depthOfSearchTree);
    printf("SIZE OF SEARCH TREE %lld\n", sizeOfSearchTree);
    searchtree.resize(sizeOfSearchTree);

    // Wrong since numEdges < neighbors (up to double the num edges, in and out)
    //hipMalloc(&dedgestatus, sizeof(int)*graph.nrEdges) != hipSuccess || 
    if (hipMalloc(&dedgestatus, sizeof(int)*graph.neighbours.size()) != hipSuccess || 
        hipMalloc(&dedges, sizeof(mtc::Edge)*graph.nrEdges) != hipSuccess || 
        hipMalloc(&dlength, sizeof(int)*graph.nrVertices) != hipSuccess || 
        hipMalloc(&dsearchtree, sizeof(int2)*sizeOfSearchTree) != hipSuccess || 
        hipMalloc(&duncoverededges, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dfullpathcount, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dnumleaves, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dremainingedges, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&dnumberofdynamicallyaddedvertices, sizeof(int)*1) != hipSuccess || 
        hipMalloc(&ddynamicallyaddedvertices_csr, sizeof(int)*(depthOfSearchTree+1)) != hipSuccess || 
        hipMalloc(&ddynamicallyaddedvertices, sizeof(int)*(kPrime)) != hipSuccess ||        
        hipMalloc(&dfinishedLeavesPerLevel, sizeof(float)*(depthOfSearchTree+1)) != hipSuccess)
	{
		cerr << "Not enough memory on device!" << endl;
		throw exception();
	}
    dmtch.resize(graph.nrVertices);
    dmatch = thrust::raw_pointer_cast(&dmtch[0]);

    edgestatus.resize(graph.neighbours.size());
    newdegrees.resize(graph.nrVertices);
    dynamcverts.resize(graph.nrVertices);

    hipMemcpy(dedges, &graph.edges[0], sizeof(mtc::Edge)*graph.nrEdges, hipMemcpyHostToDevice);

    // Since these are 32 byte sets, simply double for int2
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dsearchtree),  0, size_t(2*sizeOfSearchTree));

    ReinitializeArrays();
	hipChannelFormatDesc neighbourRangesTextureDesc = hipCreateChannelDesc<int2>();

	neighbourRangesTexture.addressMode[0] = hipAddressModeWrap;
	neighbourRangesTexture.filterMode = hipFilterModePoint;
	neighbourRangesTexture.normalized = false;
	hipBindTexture(0, neighbourRangesTexture, (void *)matcher.dneighbourRanges, neighbourRangesTextureDesc, sizeof(int2)*graph.neighbourRanges.size());
	
	hipChannelFormatDesc neighboursTextureDesc = hipCreateChannelDesc<int>();

	neighboursTexture.addressMode[0] = hipAddressModeWrap;
	neighboursTexture.filterMode = hipFilterModePoint;
	neighboursTexture.normalized = false;
	hipBindTexture(0, neighboursTexture, (void *)matcher.dneighbours, neighboursTextureDesc, sizeof(int)*graph.neighbours.size());


}

VCGPU::~VCGPU(){
    if(solutionCantExist){
        hipFree(ddegrees);
        hipFree(dsolution);
        hipFree(dremainingedges);
        hipFree(dsizeofkernelsolution);
    } else {
        hipFree(dedgestatus);
        hipFree(dedges);
        hipFree(dlength);
        hipFree(dsearchtree);
        hipFree(duncoverededges);
        hipFree(dfullpathcount);
        hipFree(dnumleaves);
        hipFree(dremainingedges);
        hipFree(dnumberofdynamicallyaddedvertices);
        hipFree(ddynamicallyaddedvertices_csr);
        hipFree(ddynamicallyaddedvertices);
        hipFree(dfinishedLeavesPerLevel);
        hipFree(dsolution);
        hipFree(ddegrees);
        printf("Finished deallocating memory\n");
        hipUnbindTexture(neighboursTexture);
        hipUnbindTexture(neighbourRangesTexture);
    }
}

long long VCGPU::CalculateSpaceForDesiredNumberOfLevels(int NumberOfLevels){
    long long summand= 0;
    // ceiling(vertexCount/2) loops
    for (int i = 0; i <= NumberOfLevels; ++i){
        summand += pow (3.0, i);
        finishedLeavesPerLevel[i] = i ? 0 : 1;
        totalLeavesPerLevel[i] = pow (3.0, i);
    }
    return summand;
}

void VCGPU::GetDeviceVectors(int nrVertices, std::vector<int> & fll, std::vector<int> & bll, std::vector<int> & length)
{
	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&fll[0], dforwardlinkedlist, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess ||
		hipMemcpy(&bll[0], dbackwardlinkedlist, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess ||
        hipMemcpy(&length[0], dlength, sizeof(int)*nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}
}
void VCGPU::GetLengthStatistics(int nrVertices, int threadsPerBlock, int *dbackwardlinkedlist, int *dlength, int *dreducedlength)
{
	int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
	ReducePathLengths<<<blocksPerGrid, threadsPerBlock>>>(nrVertices, dbackwardlinkedlist, dlength, dreducedlength);
}

int4 VCGPU::numberCompletedPaths(int nrVertices, 
                        int leafIndex,
                        int depthOfLeaf,
                        int *dbackwardlinkedlist, 
                        int *dlength,
                        int recursiveStackDepth){
	int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    PopulateSearchTree<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            sizeOfSearchTree, 
                                                            depthOfSearchTree,
                                                            leafIndex,
                                                            dfinishedLeavesPerLevel,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist, 
                                                            dlength,
                                                            dfullpathcount,
                                                            dsearchtree);
    // Dont bother with looking for pendants if I'm out of space.
    if (numberofdynamicallyaddedvertices<kPrime){
        DetectAndSetPendantPathsCase3<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            kPrime,
                                                            dmatch,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist,
                                                            dedgestatus, 
                                                            dlength,
                                                            dnumberofdynamicallyaddedvertices,
                                                            ddynamicallyaddedvertices);
        DetectAndSetPendantPathsCase4<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            kPrime,
                                                            dmatch,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist,
                                                            dedgestatus, 
                                                            dlength,
                                                            dnumberofdynamicallyaddedvertices,
                                                            ddynamicallyaddedvertices);
    }
    // Create CSR entry for dynamically added verts
    hipMemcpy(&ddynamicallyaddedvertices_csr[recursiveStackDepth], dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToDevice);


    hipMemcpy(&fullpathcount, &dfullpathcount[0], sizeof(int)*1, hipMemcpyDeviceToHost);
    //hipMemcpy(&numberofdynamicallyaddedvertices, &dnumberofdynamicallyaddedvertices[0], sizeof(int)*1, hipMemcpyDeviceToHost);

    int4 myActiveLeaves = CalculateLeafOffsets(leafIndex,
                                                fullpathcount);


    //printf("My active leaves %d %d %d %d\n", myActiveLeaves.x, myActiveLeaves.y, myActiveLeaves.z, myActiveLeaves.w);
    return myActiveLeaves;
}

int4 VCGPU::numberCompletedPathsTest(int nrVertices, 
                        int leafIndex,
                        int *dbackwardlinkedlist, 
                        int *dlength,
                        int recursiveStackDepth){

    fullpathcount = 1+rand()%20;

    //hipMemcpy(&dfullpathcount[0], &fullpathcount, sizeof(int)*1, hipMemcpyHostToDevice);

	int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    PopulateSearchTreeTest<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            sizeOfSearchTree, 
                                                            depthOfSearchTree,
                                                            leafIndex,
                                                            dfinishedLeavesPerLevel,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist, 
                                                            dlength,
                                                            dfullpathcount,
                                                            dsearchtree,
                                                            fullpathcount);

    hipMemcpy(&fullpathcount, &dfullpathcount[0], sizeof(int)*1, hipMemcpyDeviceToHost);
    //hipMemcpy(&numberofdynamicallyaddedvertices, &dnumberofdynamicallyaddedvertices[0], sizeof(int)*1, hipMemcpyDeviceToHost);

    int4 myActiveLeaves = CalculateLeafOffsets(leafIndex,
                                                fullpathcount);


    //printf("My active leaves %d %d %d %d\n", myActiveLeaves.x, myActiveLeaves.y, myActiveLeaves.z, myActiveLeaves.w);
    return myActiveLeaves;
}

// Initial kernelization before search tree is built
void VCGPU::bussKernelizationP1(){
    hipMemcpy(ddegrees, graph.degrees.data(), sizeof(int)*graph.nrVertices, hipMemcpyHostToDevice);
    int blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    BussKernelizationP1Kernel<<<blocksPerGrid, threadsPerBlock>>>(graph.nrVertices, 
                                                                k, 
                                                                ddegrees,
                                                                dsolution,
                                                                dsizeofkernelsolution);
    hipMemcpy(&sizeOfKernelSolution, dsizeofkernelsolution, sizeof(int)*1, hipMemcpyDeviceToHost);
    hipMemcpy(&solution[0], dsolution, sizeof(int)*sizeOfKernelSolution, hipMemcpyDeviceToHost);
}

// Initial kernelization before search tree is built
void VCGPU::bussKernelizationP2(){
    printf("Remaining edges before Kernel %d\n", graph.nrEdges);
    // Using the indices to calculate degrees requires doubling and then halving
    // Since each edge is counted twice, once in each connecting vertex's indices.x to indices.y
    remainingedges = 2*graph.nrEdges;
    hipMemcpy(dremainingedges, &remainingedges, sizeof(int)*1, hipMemcpyHostToDevice);
    int blocksPerGrid = (sizeOfKernelSolution + threadsPerBlock - 1)/threadsPerBlock;
    //printf("Launching %d blocks for a solution of size %d\n", blocksPerGrid, sizeOfKernelSolution);
    BussKernelizationP2Kernel<<<blocksPerGrid, threadsPerBlock>>>(sizeOfKernelSolution,
                                                                ddegrees,
                                                                dremainingedges,
                                                                dsolution);
    hipMemcpy(&remainingedges, dremainingedges, sizeof(int)*1, hipMemcpyDeviceToHost);
    // Using the indices to calculate degrees requires doubling and then halving
    // Since each edge is counted twice, once in each connecting vertex's indices.x to indices.y
    remainingedges/=2;
    printf("Remaining edges after Kernel %d\n", remainingedges);
}

void VCGPU::bussKernelizationP1(int root, int recursiveStackDepth, bool & foundSolution){

}


/*
void VCGPU::eraseDynVertsOfRecursionLevel(int recursiveStackDepth){

    hipMemcpy(&numberofdynamicallyaddedverticesLB, ddynamicallyaddedvertices_csr[recursiveStackDepth], sizeof(int)*1, hipMemcpyDeviceToHost);
    hipMemcpy(&numberofdynamicallyaddedverticesUB, ddynamicallyaddedvertices_csr[recursiveStackDepth+1], sizeof(int)*1, hipMemcpyDeviceToHost);
    int numberToErase = numberofdynamicallyaddedverticesUB - numberofdynamicallyaddedverticesLB;

    hipMemcpy(&numberofdynamicallyaddedvertices, dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);
    numberofdynamicallyaddedvertices -= numberToErase;
    hipMemcpy(&dnumberofdynamicallyaddedvertices, numberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);

}
*/
// 2 Possibilities for recycling the paths of length 1&2
// Depending on whether we want to perform parallel frontier splitting.

// PFS (NO) - this is my first implementation.
// 1) Continue matching from a leaf, after removing edges 
// of included vertices and un-coloring the non-included vertices
// This approach will prioritize DF growth of the search tree.
// This allows us to only maintain 1 copy of the edge status in GPU mem.

// PFS (YES) - can try this in second implementation.
// 2) Evaluate each path for pendantness at each leaf node.
// This requires storing the edge status of each leaf node
// for any benefit to be seen.

//  However, it is MUCH easier to just wipe away intermediate paths
// And perform BFS at each leaf node, if we stick to BFS for as 
// long as complete levels can be formed.

// Will most likely copy back the frontier bool array and iterate through each frontier one at a time
// for v1.
//for (int activeRoot = leftMostLeafOfLevel; activeRoot < rightMostLeafOfLevel; ++activeRoot){

// For DFS, we'd assume that every level beneath the last BFS has
// started at its left most child, and will need to be recursively
// searched from the bottom.

// It's important that this method is called for the leaf
// nodes, but that number of new leaves == 0
// so the pendant paths are found if any remain.
void VCGPU::FindCover(int root,
                      int recursiveStackDepth,
                      bool & foundSolution){
    if (foundSolution)
        return;

	int blocksPerGrid = (graph.nrEdges + threadsPerBlock - 1)/threadsPerBlock;
    int depthOfLeaf = ceil(logf(2*root + 1) / logf(3)) - 1;

    /*
    int depthOfLeaf;
    if (root)
        depthOfLeaf = ceil(logf(2*root + 1) / logf(3)) - 1;
    else
        depthOfLeaf = root;
    */

    #ifndef NDEBUG
    printf("Called FindCover li %d rl %d \n", root, recursiveStackDepth);
    printf("depthOfLeaf %d depthOfSearchTree %d\n",  depthOfLeaf, depthOfSearchTree);
    #endif    
    hipMemcpy(&finishedLeavesPerLevel[1], &dfinishedLeavesPerLevel[1], sizeof(float)*depthOfSearchTree, hipMemcpyDeviceToHost);

    curs_set (0);
    for(int i = 0; i <= depthOfSearchTree; ++i){
        mvprintw (i, 4, "Depth %d %f Complete %f/%f\n", i, finishedLeavesPerLevel[i]/totalLeavesPerLevel[i], finishedLeavesPerLevel[i], totalLeavesPerLevel[i]);
    }
    refresh ();
    if (depthOfLeaf > depthOfSearchTree){
        return;
    }

    // If you want to check the quality of each match, uncomment
    // Else, the only noticable changes will be in the recursion stack 
    // and the device search tree.
    //std::vector<int> match;
    //matcher.initialMatching(match);

//    printf("\033[A\33[2K\rCalling Find Cover from %d, level depth of leaf %d\n", root, depthOfLeaf);
    numoftreeverts = 2*(depthOfLeaf);
    if (sizeOfKernelSolution+numoftreeverts+numofdynamcverts < k) {
        ReinitializeArrays();
        hipDeviceSynchronize();
        // TODO - Need to set the pendant vertices also.
        SetEdgesOfLeaf(root);

        // Test algebra, comment Match(root)
        Match(root);
        hipDeviceSynchronize();

        //matcher.copyMatchingBackToHost(match);
        // Need to pass device pointer to LOP
        // Test algebra, use Test
        // Might have an error if 1 single path found.
        hipMemcpy(&numberofdynamicallyaddedvertices, &dnumberofdynamicallyaddedvertices[0], sizeof(int)*1, hipMemcpyDeviceToHost);
        int4 newLeaves = numberCompletedPaths(graph.nrVertices, root, depthOfLeaf, dbackwardlinkedlist, dlength, recursiveStackDepth);
        hipMemcpy(&numberofdynamicallyaddedvertices, &dnumberofdynamicallyaddedvertices[0], sizeof(int)*1, hipMemcpyDeviceToHost);

        //int4 newLeaves = numberCompletedPathsTest(graph.nrVertices, root, dbackwardlinkedlist, dlength, recursiveStackDepth);
        hipDeviceSynchronize();
        hipMemcpy(&finishedLeavesPerLevel[1], &dfinishedLeavesPerLevel[1], sizeof(float)*(depthOfSearchTree), hipMemcpyDeviceToHost);

        curs_set (0);
        for(int i = 0; i <= depthOfSearchTree; ++i){
            mvprintw (i, 4, "Depth %d %f Complete %f/%f\n", i, finishedLeavesPerLevel[i]/totalLeavesPerLevel[i], finishedLeavesPerLevel[i], totalLeavesPerLevel[i]);
        }
        refresh ();
        if (root != 0){
            hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(duncoverededges),  0, size_t(1));
            hipDeviceSynchronize();
            EvaluateSingleLeafNode<<<blocksPerGrid, threadsPerBlock, (sizeOfKernelSolution + 2*(depthOfLeaf+1))*sizeof(int)>>>(
                                                                                                graph.nrEdges,
                                                                                                root,
                                                                                                depthOfLeaf,
                                                                                                sizeOfKernelSolution,
                                                                                                dsolution,
                                                                                                dedges, 
                                                                                                dsearchtree,
                                                                                                dnumberofdynamicallyaddedvertices,
                                                                                                ddynamicallyaddedvertices,
                                                                                                duncoverededges);
            hipMemcpy(&uncoverededges, duncoverededges, sizeof(int)*1, hipMemcpyDeviceToHost);
            if (uncoverededges){
                #ifndef NDEBUG
                printf("leaf index %d is not a solution :  %d edges are uncovered\n", root, uncoverededges);
                #endif
            } else {   
                printf("leaf index %d is a solution :  %d edges are uncovered\n", root, uncoverededges);
                FillSolutionArray<<<1,1>>>(root,
                                    dsolution,
                                    sizeOfKernelSolution,
                                    dsearchtree,
                                    dnumberofdynamicallyaddedvertices,
                                    ddynamicallyaddedvertices);

                hipMemcpy(&numofdynamcverts, dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);
                

                numoftreeverts = 2*(depthOfLeaf);
                printf("sizeOfKernelSolution %d numoftreeverts %d numofdynamcverts %d\n", sizeOfKernelSolution,numoftreeverts,numofdynamcverts);
                solutionSize = sizeOfKernelSolution+numoftreeverts+numofdynamcverts;
                hipMemcpy(solution.data(), dsolution, sizeof(int)*solutionSize, hipMemcpyDeviceToHost);

                foundSolution = true;
                hipDeviceSynchronize();
                checkLastErrorCUDA(__FILE__, __LINE__);    
                hipMemcpy(&remainingedges, dremainingedges, sizeof(int)*1, hipMemcpyDeviceToHost);
                hipMemcpy(&edgestatus[0], dedgestatus, sizeof(int)*graph.neighbours.size(), hipMemcpyDeviceToHost);
                hipMemcpy(&newdegrees[0], ddegrees, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
                hipMemcpy(&searchtree[0], dsearchtree, sizeof(int2)*searchtree.size(), hipMemcpyDeviceToHost);
                hipMemcpy(&numofdynamcverts, dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);
                hipMemcpy(&dynamcverts[0], ddynamicallyaddedvertices, sizeof(int)*numofdynamcverts, hipMemcpyDeviceToHost);

                Gviz.DrawInputGraphColored(graph,
                                        root,
                                        searchtree,
                                        numofdynamcverts,
                                        dynamcverts,
                                        dmtch,
                                        dfll,
                                        dbll,
                                        root);
                Gviz.DrawSearchTree(sizeOfSearchTree,
                                &searchtree[0],
                                root);  

                return;
            }
        }

        #ifndef NDEBUG
        hipMemcpy(&remainingedges, dremainingedges, sizeof(int)*1, hipMemcpyDeviceToHost);
        hipMemcpy(&edgestatus[0], dedgestatus, sizeof(int)*graph.neighbours.size(), hipMemcpyDeviceToHost);
        hipMemcpy(&newdegrees[0], ddegrees, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost);
        hipMemcpy(&searchtree[0], dsearchtree, sizeof(int2)*searchtree.size(), hipMemcpyDeviceToHost);
        hipMemcpy(&numofdynamcverts, dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);
        hipMemcpy(&dynamcverts[0], ddynamicallyaddedvertices, sizeof(int)*numofdynamcverts, hipMemcpyDeviceToHost);

        Gviz.DrawInputGraphColored(graph,
                                root,
                                searchtree,
                                numofdynamcverts,
                                dynamcverts,
                                dmtch,
                                dfll,
                                dbll,
                                root);
        Gviz.DrawSearchTree(sizeOfSearchTree,
                        &searchtree[0],
                        root);   
            
        printf("leaf index %d LO %d %d LO %d %d\n", root, newLeaves.x, newLeaves.y, newLeaves.z, newLeaves.w);
        #endif
        while(newLeaves.x < newLeaves.y && newLeaves.x < sizeOfSearchTree){
            FindCover(newLeaves.x, recursiveStackDepth+1, foundSolution);
            ++newLeaves.x;
        }
        depthOfLeaf = ceil(logf(2*newLeaves.z + 1) / logf(3)) - 1;
        while(newLeaves.z < newLeaves.w && newLeaves.z < sizeOfSearchTree){
            FindCover(newLeaves.z, recursiveStackDepth+1, foundSolution);
            ++newLeaves.z;
        }
    }
    //PrintData (); 
    // Wipe away my pendant nodes from shared list
    eraseDynVertsOfRecursionLevel<<<1, threadsPerBlock>>>(recursiveStackDepth,
                                              dnumberofdynamicallyaddedvertices, 
                                              ddynamicallyaddedvertices_csr, 
                                              ddynamicallyaddedvertices);
    hipMemcpy(&numofdynamcverts, dnumberofdynamicallyaddedvertices, sizeof(int)*1, hipMemcpyDeviceToHost);

}

void VCGPU::CallDrawSearchTree(std::string prefix){
    hipMemcpy(&searchtree[0], dsearchtree, sizeof(int2)*searchtree.size(), hipMemcpyDeviceToHost);
    Gviz.DrawSearchTree(sizeOfSearchTree,
					&searchtree[0],
					prefix); 
}



void VCGPU::SetEdgesOfLeaf(int leafIndex){
    // Root of search tree is empty.
    if (leafIndex == 0)
        return;
	int blocksPerGrid = 2*(ceil(logf(2*leafIndex + 1) / logf(3)) - (int)(leafIndex==0));
    SetEdges<<<blocksPerGrid, threadsPerBlock>>>(leafIndex,
                                                dedgestatus,
                                                ddegrees,
                                                dsearchtree);
	blocksPerGrid = (graph.nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    CalculateDegrees<<<blocksPerGrid, threadsPerBlock>>>(graph.nrVertices,
                                                dedgestatus,
                                                ddegrees);
    hipDeviceSynchronize();
    checkLastErrorCUDA(__FILE__, __LINE__);
}

void VCGPU::PrintData (){
   
    printf("neighbours size %d num edges %d\n",  graph.neighbours.size(), graph.nrEdges);
    printf("Row Offs\n");
    for (int i = 0; i < graph.nrVertices; ++i){
        printf("%d %d, ",graph.neighbourRanges[i].x, graph.neighbourRanges[i].y);
    }
    printf("\n");
    printf("Cols\n");
    for (int i = 0; i < graph.neighbours.size(); ++i){
        printf("%d ",graph.neighbours[i]);
    }
    printf("\n");
    printf("Vals\n");
    for (int i = 0; i < graph.neighbours.size(); ++i){
        printf("%d ",edgestatus[i]);
    }
    printf("\n");
    printf("Degrees\n");
    for (int i = 0; i < graph.nrVertices+1; ++i){
        printf("%d ", newdegrees[i]);
    }
    printf("Remaining Edges\n");
    printf("%d\n", remainingedges);
    printf("\n");
    printf("Dyn Verts\n");
    for (int i = 0; i < numofdynamcverts; ++i){
        printf("%d ", dynamcverts[i]);
    }

}
void VCGPU::Match(int leafIndex){
    //Initialise timers.
    hipEvent_t t0, t1, t2, t3;
    float time0, time1;

    hipEventCreate(&t0);
    hipEventCreate(&t1);
    hipEventCreate(&t2);
    hipEventCreate(&t3);
    
    hipEventRecord(t0, 0);
    hipEventSynchronize(t0);

    matcher.performMatching(dmatch, t1, t2, dforwardlinkedlist, dbackwardlinkedlist, dlength, dsearchtree, ddynamicallyaddedvertices, dnumberofdynamicallyaddedvertices, sizeOfKernelSolution, leafIndex);
    
    hipEventElapsedTime(&time1, t1, t2);
    hipEventRecord(t3, 0);
    hipEventSynchronize(t3);
    //Measure the total elapsed time (including data transfer) and the calculation time.
    hipEventElapsedTime(&time0, t0, t3);
    hipEventElapsedTime(&time1, t1, t2);
    //Destroy timers.
    hipEventDestroy(t3);
    hipEventDestroy(t2);
    hipEventDestroy(t1);
    hipEventDestroy(t0);
}

void VCGPU::ReinitializeArrays(){
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dedgestatus),  1, size_t(graph.neighbours.size()));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dlength),  0, size_t(graph.nrVertices));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dfullpathcount),  0, size_t(1));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dnumleaves),  0, size_t(1));
    hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(dremainingedges),  0, size_t(1));
    // Only >= 0 are heads of full paths
    // Before implementing recursive backtracking, I can keep performing this memcpy to set degrees
    // and the remove tentative vertices to check a cover.
    hipMemcpy(ddegrees, &graph.degrees[0], sizeof(int)*graph.nrVertices, hipMemcpyHostToDevice);

    dfll.clear();
    dfll.resize(graph.nrVertices);
	thrust::sequence(dfll.begin(),dfll.end(), 0, 1);
	dforwardlinkedlist = thrust::raw_pointer_cast(&dfll[0]);
	
    dbll.clear();
    dbll.resize(graph.nrVertices);
	thrust::sequence(dbll.begin(),dbll.end(), 0, 1);
	dbackwardlinkedlist = thrust::raw_pointer_cast(&dbll[0]);

}

__global__ void ReduceEdgeStatusArray(int nrNeighbors,
							int *dedgestatus,
                            int* dremainingedges){
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int val = 0;
    unsigned mask = 0xFFFFFFFFU;
    int lane = threadIdx.x % warpSize;
    int warpID = threadIdx.x / warpSize;
    while (idx < nrNeighbors) { // grid stride loop to load
        val += dedgestatus[idx];
        idx += gridDim.x*blockDim.x;
    }
    // 1st warp-shuffle reduction
    for (int offset = warpSize/2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(mask, val, offset);
    if (lane == 0) sdata[warpID] = val;
    __syncthreads(); // put warp results in shared mem
    // hereafter, just warp 0
    if (warpID == 0){
        // reload val from shared mem if warp existed
        val = (tid < blockDim.x/warpSize)?sdata[lane]:0;
        // final warp-shuffle reduction
        for (int offset = warpSize/2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(mask, val, offset);
        if (tid == 0) atomicAdd(dremainingedges, val);
    }
}

// Only launch 1 block so sync threads prevents retrieving a bad UB
__global__ void eraseDynVertsOfRecursionLevel(int recursiveStackDepth,
                                              int * dnumberofdynamicallyaddedvertices, 
                                              int * ddynamicallyaddedvertices_csr, 
                                              int * ddynamicallyaddedvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int LB = ddynamicallyaddedvertices_csr[recursiveStackDepth];
    int UB = ddynamicallyaddedvertices_csr[recursiveStackDepth+1];
    for (int entry = LB + threadID; entry < UB; entry += blockDim.x)
        ddynamicallyaddedvertices[entry] = 0;

    __syncthreads();
    if (threadID == 0){
        dnumberofdynamicallyaddedvertices -= (UB - LB);
        ddynamicallyaddedvertices_csr[recursiveStackDepth+1] = 0;
    }
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PopulateSearchTree(int nrVertices, 
                                    int sizeOfSearchTree,
                                    int depthOfSearchTree,
                                    int leafIndex,
                                    float * dfinishedLeavesPerLevel,
                                    int *dforwardlinkedlist, 
                                    int *dbackwardlinkedlist, 
                                    int *dlength, 
                                    int *dfullpathcount,
                                    int2* dsearchtree){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 3 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = threadID;
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];
    int fourth = dforwardlinkedlist[third];

    int leavesToProcess = atomicAdd(&dfullpathcount[0], 1) + 1;
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // r = 3, a = 1, solve for n given s_n = leavesToProcess ∈ [1,m/4]
    // where m = number of vertices.
    // s_n = (1-r^(n+1))/(1-r)
    // s_n * (1-3) = -2*s_n = (1-r^(n+1))
    //     = -2*s_n - 1 = -3^(n+1)
    //     =  2*s_n + 1  =  3^(n+1)
    //     =  log(2*s_n + 1) = n+1*log(3)
    //     =  log(2*s_n + 1)/log(3) = n + 1
    //     =  log(2*s_n + 1)/log(3) - 1 = n
    // n is the number of terms in the closed form solution.
    // Alternatively, n is the number of levels in the search tree.
    int n = ceil(logf(2*leavesToProcess + 1) / logf(3));
    //float nf = logf(2*leavesToProcess + 1) / logf(3);
    int arbitraryParameter = 3*((3*leafIndex)+1);
    // At high powers, the error of transendental powf causes bugs.
    //int leftMostLeafIndexOfIncompleteLevel = ((2*arbitraryParameter+3)*powf(3.0, n-1) - 3)/6;

    // Discrete calculation without trancendentals
    int leftMostLeafIndexOfIncompleteLevel = (2*arbitraryParameter+3);
    int multiplicand = 1;
    for (int i = 1; i < n; ++i)
        multiplicand*=3;
    leftMostLeafIndexOfIncompleteLevel*=multiplicand;
    leftMostLeafIndexOfIncompleteLevel-=3;
    leftMostLeafIndexOfIncompleteLevel/=6;

    int treeSizeNotIncludingThisLevel = (1.0 - multiplicand)/(1.0 - 3.0); 
    // At high powers, the error of transendental powf causes bugs. 
    //int treeSizeNotIncludingThisLevel = (1.0 - powf(3.0, ((n+1)-1)))/(1.0 - 3.0);  
    // Test from root for now, this code can have an arbitrary root though
    //leafIndex = global_active_leaves[globalIndex];
//    leafIndex = 0;
    // Closed form solution of recurrence relation shown in comment above method
    // Subtract 1 because reasons???
    int internalLeafIndex = leavesToProcess - 1 - treeSizeNotIncludingThisLevel;
    //int internalLeafIndex = leavesToProcess - treeSizeNotIncludingThisLevel;
    int levelOffset = leftMostLeafIndexOfIncompleteLevel + 3*internalLeafIndex;

    if (sizeOfSearchTree <= levelOffset ||
        sizeOfSearchTree <= (levelOffset + 1) ||
        sizeOfSearchTree <= (levelOffset +2)||
        levelOffset< 0 ||
        (levelOffset+1) < 0 ||
        (levelOffset + 2)< 0){
            atomicSub(&dfullpathcount[0], 1);
            //printf("child %d exceeded srch tree depth\n", levelOffset);
            return;
    }
    // Add to device pointer of level
    int depthOfLeaf = floor(logf(2*levelOffset + 1) / logf(3));
    //printf("leafIndex %d atomicAdd(&dfinishedLeavesPerLevel[%d], 3) newleaves %d - %d\n", leafIndex, depthOfLeaf,levelOffset, levelOffset + 2); 
    atomicAdd(&dfinishedLeavesPerLevel[depthOfLeaf], 3.0); 
    // Test from root for now, this code can have an arbitrary root though
    dsearchtree[levelOffset + 0] = make_int2(first, third);
    dsearchtree[levelOffset + 1] = make_int2(second, third);
    dsearchtree[levelOffset + 2] = make_int2(second, fourth);   
}


// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PopulateSearchTreeTest(int nrVertices, 
                                    int sizeOfSearchTree,
                                    int depthOfSearchTree,
                                    int leafIndex,
                                    float * dfinishedLeavesPerLevel,
                                    int *dforwardlinkedlist, 
                                    int *dbackwardlinkedlist, 
                                    int *dlength, 
                                    int *dfullpathcount,
                                    int2* dsearchtree,
                                    int fullpathcount){
	const unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= fullpathcount) 
            return;

    unsigned int leavesToProcess = atomicAdd(&dfullpathcount[0], 1) + 1;
    // https://en.wikipedia.org/wiki/Geometric_series#Closed-form_formula
    // r = 3, a = 1, solve for n given s_n = leavesToProcess ∈ [1,m/4]
    // where m = number of vertices.
    // s_n = (1-r^(n+1))/(1-r)
    // s_n * (1-3) = -2*s_n = (1-r^(n+1))
    //     = -2*s_n - 1 = -3^(n+1)
    //     = 2*s_n + 1  =  3^(n+1)
    //     = log(2*s_n + 1) = n+1*log(3)
    //     = log(2*s_n + 1)/log(3) = n + 1
    //     = log(2*s_n + 1)/log(3) - 1 = n
    // n is the number of terms in the closed form solution.
    // Alternatively, n is the number of levels in the search tree.
    unsigned int n = ceil(logf(2*leavesToProcess + 1) / logf(3));
    //float nf = logf(2*leavesToProcess + 1) / logf(3);
    unsigned int arbitraryParameter = 3*((3*leafIndex)+1);
    // At high powers, the error of transendental powf causes bugs.
    //unsigned int leftMostLeafIndexOfIncompleteLevel = ((2*arbitraryParameter+3)*powf(3.0, n-1) - 3)/6;

    // Discrete calculation without trancendentals
    unsigned int leftMostLeafIndexOfIncompleteLevel = (2*arbitraryParameter+3);
    unsigned int multiplicand = 1;
    for (unsigned int i = 1; i < n; ++i)
        multiplicand*=3;
    leftMostLeafIndexOfIncompleteLevel*=multiplicand;
    leftMostLeafIndexOfIncompleteLevel-=3;
    leftMostLeafIndexOfIncompleteLevel/=6;

    unsigned int treeSizeNotIncludingThisLevel = (1.0 - multiplicand)/(1.0 - 3.0); 
    // At high powers, the error of transendental powf causes bugs. 
    //unsigned int treeSizeNotIncludingThisLevel = (1.0 - powf(3.0, ((n+1)-1)))/(1.0 - 3.0);  
    // Test from root for now, this code can have an arbitrary root though
    //leafIndex = global_active_leaves[globalIndex];
//    leafIndex = 0;
    // Closed form solution of recurrence relation shown in comment above method
    // Subtract 1 because reasons???
    unsigned int internalLeafIndex = leavesToProcess - 1 - treeSizeNotIncludingThisLevel;
    //unsigned int internalLeafIndex = leavesToProcess - treeSizeNotIncludingThisLevel;
    unsigned int levelOffset = leftMostLeafIndexOfIncompleteLevel + 3*internalLeafIndex;

    if (sizeOfSearchTree <= levelOffset ||
        sizeOfSearchTree <= (levelOffset + 1) ||
        sizeOfSearchTree <= (levelOffset +2)||
        levelOffset< 0 ||
        (levelOffset+1) < 0 ||
        (levelOffset + 2)< 0){
            atomicSub(&dfullpathcount[0], 1);
            //printf("child %d exceeded srch tree depth\n", levelOffset);
            return;
    }
    // Add to device pointer of level
    //atomicAdd(&dfinishedLeavesPerLevel[depthOfLeaf], 3); 
    // Test from root for now, this code can have an arbitrary root though
    dsearchtree[levelOffset + 0] = make_int2(levelOffset, levelOffset);
    dsearchtree[levelOffset + 1] = make_int2(levelOffset + 1, levelOffset + 1);
    dsearchtree[levelOffset + 2] = make_int2(levelOffset + 2, levelOffset + 2);   
}

// Each thread will take an edge.  Each thread will loop through the answer
// until it finds either vertex a or b of an edge.
// if it reaches the end of the answer without terminating, it isn't a solution.
// Amount of shared memory should be 2*depth*sizeof(unsigned int)
// 
__global__ void EvaluateSingleLeafNode(int nrEdges,
                                    int leafIndex,
                                    int depthOfLeaf,
                                    int sizeOfKernelSolution,
                                    int * dsolution,
                                    mtc::Edge * dedges, 
                                    int2 * dsearchtree,
                                    int * dnumberofdynamicallyaddedvertices,
                                    int * ddynamicallyaddedvertices,
                                    int * uncoverededges){
    extern __shared__ int soln[];
	const int edgeID = blockIdx.x*blockDim.x + threadIdx.x;
    if (edgeID >= nrEdges)
        return;
    int UBDyn = dnumberofdynamicallyaddedvertices[0];
    const int tid = threadIdx.x;
    #ifndef NDEBUG
    if (tid == 0){
        printf("depthOfLeaf %d\n", depthOfLeaf);
        printf("UBDyn %d\n", UBDyn);
    }
    #endif
    int leafIndexSoln = leafIndex;
    int2 nodeEntry;
    int counter = 0;
    int i;
    // Load solution into shared memory
    for(i = threadIdx.x; i < sizeOfKernelSolution; i+=blockDim.x){
        soln[i] = dsolution[i];
    }
    // Currently the same first sizeOfKernelSolution threads
    // have to first copy the kernel solution
    // then copy the tree solution.
    // This can occur in parallel with fancy for loop indexing.
    if (tid <= depthOfLeaf){
        for(int i = 0; i < tid; ++i){
            if(leafIndexSoln % 3 == 0){
                --leafIndexSoln;
                leafIndexSoln = leafIndexSoln / 3;
            } else {
                leafIndexSoln = leafIndexSoln / 3;
            }
        }
        nodeEntry = dsearchtree[leafIndexSoln];
        soln[sizeOfKernelSolution + 2*tid] = nodeEntry.x;
        soln[sizeOfKernelSolution + 2*tid + 1] = nodeEntry.y;
        #ifndef NDEBUG
        printf("tid %d is adding %d %d to soln indices %d %d \n", tid, nodeEntry.x, nodeEntry.y, 2*tid, 2*tid + 1);
        #endif
    }
        /*
        // If you decide to use sm for dynamic verts
        for (int index = 0; index < dnumberofdynamicallyaddedvertices[0]; ++index){
            soln[counter] = ddynamicallyaddedvertices[index];
            ++counter;
        }
        */
     
    __syncthreads(); // put warp results in shared mem
    #ifndef NDEBUG
    if (threadIdx.x == 0){
        printf("LI %d Tree soln\n", leafIndex);
        for (int i = 0; i < (depthOfLeaf+1)*2; ++i)
            printf("%d ", soln[i]);
        printf("\n");
    }
    __syncthreads(); // put warp results in shared mem
    #endif
    Edge & edge = dedges[edgeID];
    bool covered = false;
    for (int solutionIndex = 0; solutionIndex < sizeOfKernelSolution + (depthOfLeaf+1)*2; ++solutionIndex){
        covered |= (edge.x == soln[solutionIndex]);
        covered |= (edge.y == soln[solutionIndex]);
    }
    /*
    // If you decide to use sm for dynamic verts
    for (int solutionIndex = 0; solutionIndex < counter; ++solutionIndex){
        covered |= (edge.x == soln[solutionIndex] || edge.y == soln[solutionIndex]);
    }
    */
    #ifndef NDEBUG
    if (threadIdx.x == 0){
        printf("LI %d Dyn soln\n", leafIndex);
        for (int index = 0; index < UBDyn; ++index)
            printf("%d ", ddynamicallyaddedvertices[index]);
        printf("\n");
    }
    __syncthreads(); // put warp results in shared mem
    #endif
    for (int index = 0; index < UBDyn; ++index){
        covered |= (edge.x == ddynamicallyaddedvertices[index]);
        covered |= (edge.y == ddynamicallyaddedvertices[index]);
    }
    // Maybe do a warp shuffle and only 1 atomic add per block
    if (!covered){
        #ifndef NDEBUG
        printf("Edge (%d - %d) uncovered\n", edge.x, edge.y);
        #endif
        atomicAdd(uncoverededges, 1);
    } 
    #ifndef NDEBUG
    else {
        printf("Edge (%d - %d) covered\n", edge.x, edge.y);
    }
    #endif
}

__global__ void PrintSolutionArray(int solutionSize,
                                int * dsolution){
    for (int i = 0; i < solutionSize; ++i){
        printf("%d ", dsolution[i]);
    }
}

// Single threaded; could accelerate eventually.
__global__ void FillSolutionArray(int leafIndex,
                                int * dsolution,
                                int sizeOfKernelSolution,
                                int2 * dsearchtree,
                                int * dnumberofdynamicallyaddedvertices,
                                int * ddynamicallyaddedvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int leafIndexSoln = leafIndex;
    int2 nodeEntry;
    int counter = 0;
    int UBDyn = dnumberofdynamicallyaddedvertices[0];
    if (threadID == 0){
        while(leafIndexSoln != 0){
            nodeEntry = dsearchtree[leafIndexSoln];
            dsolution[sizeOfKernelSolution + counter] = nodeEntry.x;
            dsolution[sizeOfKernelSolution + counter + 1] = nodeEntry.y;
            //printf("Tree verts %d %d\n", nodeEntry.x, nodeEntry.y);
            if(leafIndexSoln % 3 == 0){
                --leafIndexSoln;
                leafIndexSoln = leafIndexSoln / 3;
            } else {
                leafIndexSoln = leafIndexSoln / 3;
            }
            counter += 2;
        }
        for (int index = 0; index < UBDyn; ++index){
            dsolution[sizeOfKernelSolution + counter + index] = ddynamicallyaddedvertices[index];
            //printf("Dynamic vertex %d\n", dsolution[counter+index]);
        }
    }
}
// Each block is a leaf node
// First it loads it's solution into shared memory.
// ADVANCED - If kPrime*sizeof(int) > shared memory limit, check in portions
// Each thread takes an edge, iterate over all edges,
// check of vertex a and vertex b is missing from soln
// if so, indicate in final position of shared memory.
// sync threads
// terminate prematurely if final position flag is set.
// this way we can check leaf nodes in parallel, without needing 
// an edge status array.
/*
__global__ void EvaluateLeafNodesV2(int nrEdges,
                                    mtc::Edge * dedges, 
                                    int sizeOfSearchTree,
                                    int depthOfSearchTree,
                                    int2 * dsearchtree){
    extern __shared__ int soln[];
    const int leafIndex = blockIdx.x;
    int thisThreadsSearchTreeNode;
    int2 nodeEntry;

    int tid = threadIdx.x;
    unsigned mask = 0xFFFFFFFFU;
    int lane = threadIdx.x % warpSize;
    int warpID = threadIdx.x / warpSize;
    // Load solution into shared memory
    // Need depthOfSearch minus 1 to exclude root
    for (int numberOfLevelsToAscend = threadIdx.x; numberOfLevelsToAscend < depthOfSearchTree-1; numberOfLevelsToAscend += blockDim.x){
        thisThreadsSearchTreeNode = leafIndex / pow (3.0, numberOfLevelsToAscend);
        nodeEntry = dsearchtree[thisThreadsSearchTreeNode];
        soln[2*numberOfLevelsToAscend] = nodeEntry.x;
        soln[2*numberOfLevelsToAscend + 1] = nodeEntry.y;
    }
    int covered;
    for (int e = 0; e < nrEdges; e++){
        Edge & edge = dedges[e];
        for (int vertexInAnswer = threadIdx.x; 
                    vertexInAnswer < 2*(depthOfSearchTree-1); 
                    vertexInAnswer += blockDim.x){
            covered |= edge.x == soln[vertexInAnswer] || edge.y == soln[vertexInAnswer];
        }
        // 1st warp-shuffle reduction
        for (int offset = warpSize/2; offset > 0; offset >>= 1)
            covered |= __shfl_down_sync(mask, covered, offset);
        if (lane == 0) sdata[warpID] = covered;
        __syncthreads(); // put warp results in shared mem
        // hereafter, just warp 0
        if (warpID == 0){
            // reload val from shared mem if warp existed
            val = (tid < blockDim.x/warpSize)?sdata[lane]:0;
            // final warp-shuffle reduction
            for (int offset = warpSize/2; offset > 0; offset >>= 1)
            val += __shfl_down_sync(mask, val, offset);
            if (tid == 0) atomicAdd(dremainingedges, val);
        }
    }
}
*/
// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void DetectAndSetPendantPathsCase4(int nrVertices, 
                                                int kPrime,
                                                int *match, 
                                                int *dforwardlinkedlist, 
                                                int *dbackwardlinkedlist, 
                                                int * dedgestatus,
                                                int *dlength, 
                                                int *dnumberofdynamicallyaddedvertices,
                                                int *ddynamicallyaddedvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int dynamicIndex;

	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 1 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];

    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 3){
        dynamicIndex = atomicAdd(&dnumberofdynamicallyaddedvertices[0], 1); 
        //ddynamicallyaddedvertices[dynamicIndex] = first;
        //SetEdges(first, dedgestatus);
    } else if (match[second] == 3){
        dynamicIndex = atomicAdd(&dnumberofdynamicallyaddedvertices[0], 1); 
        //ddynamicallyaddedvertices[dynamicIndex] = second;
        //SetEdges(second, dedgestatus);
    }

    if (match[first] == 3 || match[second] == 3){
        if (dynamicIndex < kPrime){
            if (match[first] == 3){
                ddynamicallyaddedvertices[dynamicIndex] = first;
            } else if (match[second] == 3){
                ddynamicallyaddedvertices[dynamicIndex] = second;
            }
            //printf("added pendant %d", dynamicIndex);
        } else {
            atomicSub(&dnumberofdynamicallyaddedvertices[0], 1);
        }
    }
}

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void DetectAndSetPendantPathsCase3(int nrVertices, 
                                              int kPrime,
                                                int *match, 
                                                int *dforwardlinkedlist, 
                                                int *dbackwardlinkedlist, 
                                                int * dedgestatus,
                                                int *dlength, 
                                                int *dnumberofdynamicallyaddedvertices,
                                                int *ddynamicallyaddedvertices){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int dynamicIndex;
    // If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 2 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];

    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 3){
        dynamicIndex = atomicAdd(&dnumberofdynamicallyaddedvertices[0], 1); 
        //ddynamicallyaddedvertices[dynamicIndex] = first;
        //SetEdges(first, dedgestatus);
    } else if (match[third] == 3){
        dynamicIndex = atomicAdd(&dnumberofdynamicallyaddedvertices[0], 1); 
        //ddynamicallyaddedvertices[dynamicIndex] = third;
        //SetEdges(third, dedgestatus);
    }

    if (match[first] == 3 || match[third] == 3){
        if (dynamicIndex < kPrime){
            if (match[first] == 3){
                ddynamicallyaddedvertices[dynamicIndex] = first;
            } else if (match[third] == 3){
                ddynamicallyaddedvertices[dynamicIndex] = third;
            }
            //printf("added pendant %d", dynamicIndex);
        } else {
            atomicSub(&dnumberofdynamicallyaddedvertices[0], 1);
        }
    }
}

// Makes sense for BFS
// For DFS use Recursive Backtracking
__global__ void GetFrontierStatus(int nrNodes,
							int *active_frontier_status){

}

__global__ void ReducePathLengths(int nrVertices,
							int *dbackwardlinkedlist,
                            int* dlength,
                            int* dreducedlength){}

__global__ void SetEdges(const int leafIndex,
                        int * dedgestatus,
                        int * ddegrees,
                        int2 *dsearchtree){

	//Determine blue and red groups using MD5 hashing.
	//Based on the Wikipedia MD5 hashing pseudocode (http://en.wikipedia.org/wiki/MD5).
	const int numberOfLevelsToAscend = blockIdx.x/2;
    //if (threadIdx.x == 0){
    int thisBlocksSearchTreeNode = leafIndex / pow (3.0, numberOfLevelsToAscend);
    //}
    int2 verticesInNode = dsearchtree[thisBlocksSearchTreeNode];
    int i;
    if (blockIdx.x % 2 == 0)
        i = verticesInNode.x;
    else 
        i = verticesInNode.y;
    int2 indices = tex1Dfetch(neighbourRangesTexture, i);
    #ifndef NDEBUG
    if (threadIdx.x == 0){
        printf("thisBlocksSearchTreeNode %d\n", thisBlocksSearchTreeNode);
        printf("Setting vertex %d\n", i);
        printf("Turning off edges between %d and %d in col array\n",indices.x,indices.y);
    }
    #endif
    for (int j = indices.x + threadIdx.x; j < indices.y; j += blockDim.x){
        //const int ni = tex1Dfetch(neighboursTexture, j);
        //printf("Turning off edge %d which is index %d of the val array\n",ni,j);
        // Set out-edges
        dedgestatus[j] = 0;
    }   
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    bool foundChild, tmp;
    // There are two possibilities for parallelization here:
    // 1) Each thread will take an out edge, and then each thread will scan the edges leaving 
    // that vertex for the original vertex.
    //for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){

    // Basically, each thread is reading wildly different data
    // 2) 1 out edge is traversed at a time, and then all the threads scan
    // all the edges leaving that vertex for the original vertex.
    // This is the more favorable data access pattern.
    const int2 indices_curr = tex1Dfetch(neighbourRangesTexture, i);
    for (int j = indices_curr.x + threadIdx.x; j < indices_curr.y; j += blockDim.x){
        const int ni = tex1Dfetch(neighboursTexture, j);    
        const int2 indices_neighbor = tex1Dfetch(neighbourRangesTexture, ni);
          for (int j_n = indices_neighbor.x; j_n < indices_neighbor.y; ++j_n){
                const int nj = tex1Dfetch(neighboursTexture, j_n);       
                foundChild = i == nj;
                // Set in-edge
                // store edge status
                tmp = dedgestatus[j_n];
                //   foundChild     tmp   (foundChild & tmp)  (foundChild & tmp)^tmp
                //1)      0          0            0                       0
                //2)      1          0            0                       0
                //3)      0          1            0                       1
                //4)      1          1            1                       0
                //
                // Case 1: isnt myChild and edge is off, stay off
                // Case 2: is myChild and edge is off, stay off
                // Case 3: isn't myChild and edge is on, stay on
                // Case 4: is myChild and edge is on, turn off
                // All this logic is necessary because we aren't using degree to set upperbound
                // we are using row offsets, which may include some edges turned off on a previous
                // pendant edge processing step.
                // Doesnt work for some reason
                // dedgestatus[j_n] ^= (foundChild & tmp);

                if(foundChild && tmp)
                    dedgestatus[j] = 0;
        }
    } 
}


// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void BussKernelizationP1Kernel(int nrVertices, 
                                        int k, 
                                        int *ddegrees,
                                        int *dsolution,
                                        int *dsizeofkernelsolution){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= nrVertices) return;
    int degree = ddegrees[threadID];
    if (degree <= k) return;
    int solutionIndex = atomicAdd(&dsizeofkernelsolution[0], 1);
    // dsolution = new int[k];
    // Prevent oob
    if (solutionIndex >= k){
        return;
    }
    dsolution[solutionIndex] = threadID;
}

__global__ void BussKernelizationP2Kernel(int sizeOfKernelSolution,
                                        int *ddegrees,
                                        int *dremainingedges,
                                        int *dsolution){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= sizeOfKernelSolution) return;
    int solnVertex = dsolution[threadID];
    int degree = ddegrees[solnVertex];
    int remainingedges = atomicSub(&dremainingedges[0], degree);
    //printf("Removed %d's %d edges : edges remaining %d\n", solnVertex, degree/2, remainingedges/2);
}

__device__ void SetEdges(   int vertexToInclude,
                            int * dedgestatus){

    int2 indices = tex1Dfetch(neighbourRangesTexture, vertexToInclude);
    for (int j = indices.x; j < indices.y; j += 1){
        //const int ni = tex1Dfetch(neighboursTexture, j);
        //printf("Turning off edge %d which is index %d of the val array\n",ni,j);
        // Set out-edges
        dedgestatus[j] = 0;
    }   
    // (u,v) is the form of edge pairs.  We are traversing over v's outgoing edges, 
    // looking for u as the destination and turning off that edge.
    bool foundChild, tmp;
    // There are two possibilities for parallelization here:
    // 1) Each thread will take an out edge, and then each thread will scan the edges leaving 
    // that vertex for the original vertex.
    //for (int edge = LB + threadIdx.x; edge < UB; edge += blockDim.x){

    // Basically, each thread is reading wildly different data
    // 2) 1 out edge is traversed at a time, and then all the threads scan
    // all the edges leaving that vertex for the original vertex.
    // This is the more favorable data access pattern.
    const int2 indices_curr = tex1Dfetch(neighbourRangesTexture, vertexToInclude);
    for (int j = indices_curr.x; j < indices_curr.y; j += 1){
        const int ni = tex1Dfetch(neighboursTexture, j);    
        const int2 indices_neighbor = tex1Dfetch(neighbourRangesTexture, ni);
          for (int j_n = indices_neighbor.x; j_n < indices_neighbor.y; ++j_n){
                const int nj = tex1Dfetch(neighboursTexture, j_n);       
                foundChild = vertexToInclude == nj;
                // Set in-edge
                // store edge status
                tmp = dedgestatus[j_n];
                //   foundChild     tmp   (foundChild & tmp)  (foundChild & tmp)^tmp
                //1)      0          0            0                       0
                //2)      1          0            0                       0
                //3)      0          1            0                       1
                //4)      1          1            1                       0
                //
                // Case 1: isnt myChild and edge is off, stay off
                // Case 2: is myChild and edge is off, stay off
                // Case 3: isn't myChild and edge is on, stay on
                // Case 4: is myChild and edge is on, turn off
                // All this logic is necessary because we aren't using degree to set upperbound
                // we are using row offsets, which may include some edges turned off on a previous
                // pendant edge processing step.
                // Doesnt work for some reason
                // dedgestatus[j_n] ^= (foundChild & tmp);

                if(foundChild && tmp)
                    dedgestatus[j] = 0;
        }
    } 
}


__global__ void CalculateDegrees(
                        int nrVertices,
                        int * dedgestatus,
                        int * ddegrees){

	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadID >= nrVertices ) return;
    int sum = 0;
    int2 indices = tex1Dfetch(neighbourRangesTexture, threadID);
    for (int j = indices.x; j < indices.y; ++j){
        sum += dedgestatus[j];
    }
    ddegrees[threadID] = sum;
}

__global__ void InitDegrees(const int nrVertices,
                            int * ddegrees){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= nrVertices) return;
    const int2 indices = tex1Dfetch(neighbourRangesTexture, i);
    ddegrees[i] = indices.y - indices.x;
}

__global__ void CalculateNumberOfLeaves(int *dfullpathcount){

}


//int leafIndex = global_active_leaf_value[leafIndex];
// Solve recurrence relation 
// g(n) = 1/6*((2*C+3)*3^n - 3)
// C depends on leafIndex
// where g(0) = left-most child of depth 1
// where g(1) = left-most child of depth 2
// where g(2) = left-most child of depth 3
// ...
//int arbitraryParameter = 3*(3*leafIndex)+1);

// currently a single root is expanded in gpu memory at a time. 
// efforts were made in the FPT-kVC "done" branch to maintain multiple copies of the graph
// and explore the search tree in parallel.

// Template this to do any type of tree
// binary, ternary, quaternary, ...
int4 CalculateLeafOffsets(              int leafIndex,
                                        int fullpathcount){

    unsigned int leavesToProcess = fullpathcount;
    unsigned int leavesFromIncompleteLvl = 1;
    unsigned int leavesFromCompleteLvl = 1;

    if (leavesToProcess == 0)
        return make_int4( leafIndex,
                          leafIndex,
                          leafIndex,
                          leafIndex);

    unsigned int n_com = floor(logf(2*leavesToProcess + 1) / logf(3));
    unsigned int n_inc = ceil(logf(2*leavesToProcess + 1) / logf(3));

    for (unsigned int i = 1; i <= n_inc; ++i)
        leavesFromIncompleteLvl*=3;

    for (unsigned int i = 1; i <= n_com; ++i)
        leavesFromCompleteLvl*=3;
    //float nf = logf(2*leavesToProcess + 1) / logf(3);
    unsigned int arbitraryParameter = 3*((3*leafIndex)+1);
    // At high powers, the error of transendental powf causes bugs.
    //unsigned int leftMostLeafIndexOfIncompleteLevel = ((2*arbitraryParameter+3)*powf(3.0, n-1) - 3)/6;

    // Discrete calculation without trancendentals
    unsigned int leftMostLeafIndexOfIncompleteLevel = (2*arbitraryParameter+3);
    unsigned int multiplicandn_inc = 1;
    for (unsigned int i = 1; i < n_inc; ++i)
        multiplicandn_inc*=3;
    leftMostLeafIndexOfIncompleteLevel*=multiplicandn_inc;
    leftMostLeafIndexOfIncompleteLevel-=3;
    leftMostLeafIndexOfIncompleteLevel/=6;

    unsigned int leftMostLeafIndexOfCompleteLevel = (2*arbitraryParameter+3);
    unsigned int multiplicandn_com = 1;
    for (unsigned int i = 1; i < n_com; ++i)
        multiplicandn_com*=3;
    leftMostLeafIndexOfCompleteLevel*=multiplicandn_com;
    leftMostLeafIndexOfCompleteLevel-=3;
    leftMostLeafIndexOfCompleteLevel/=6;

    unsigned int treeSizeNotIncludingThisLevel = (1.0 - multiplicandn_inc)/(1.0 - 3.0); 
    // At high powers, the error of transendental powf causes bugs. 
    //unsigned int treeSizeNotIncludingThisLevel = (1.0 - powf(3.0, ((n+1)-1)))/(1.0 - 3.0);  
    // Test from root for now, this code can have an arbitrary root though
    //leafIndex = global_active_leaves[globalIndex];
//    leafIndex = 0;
    // Closed form solution of recurrence relation shown in comment above method
    // Subtract 1 because reasons???
    unsigned int internalLeafIndex = leavesToProcess - 1 - treeSizeNotIncludingThisLevel;
    //unsigned int internalLeafIndex = leavesToProcess - treeSizeNotIncludingThisLevel;
    unsigned int levelOffset = leftMostLeafIndexOfIncompleteLevel + 3*internalLeafIndex;

    #ifndef NDEBUG
    printf("Leaves %d, completeLevel Level Depth %d\n",leavesToProcess, n_com);
    printf("Leaves %d, incompleteLevel Level Depth %d\n",leavesToProcess, n_inc);
    printf("Leaves %d, leavesFromCompleteLvl %d\n",leavesToProcess, leavesFromCompleteLvl);
    printf("Leaves %d, leavesFromIncompleteLvl %d\n",leavesToProcess, leavesFromIncompleteLvl);
    printf("Leaves %d, leftMostLeafIndexOfIncompleteLevel %d\n",leavesToProcess, leftMostLeafIndexOfIncompleteLevel);
    #endif
    // Grow tree leftmost first, so put the incomplete level first.
    // Shape of leaves
    //CL    -     -    o o o 
    //IL  o o o o o o
    unsigned int clb;
    unsigned int cub;

    if (n_com == n_inc){
        clb = levelOffset + 3;
        cub = levelOffset + 3;
    } else {
        clb = (levelOffset + 2)/3;
        cub = leftMostLeafIndexOfCompleteLevel + leavesFromCompleteLvl;
    }
    // Grow tree leftmost first, so put the incomplete level first.
    // Shape of leaves
    //CL    -     -    o o o 
    //IL  o o o o o o
    return make_int4(leftMostLeafIndexOfIncompleteLevel,
                     levelOffset + 3,
                     clb,
                     cub);
}

void VCGPU::CopyMatchingBackToHost(std::vector<int> & match){
	//Copy obtained matching on the device back to the host.
	if (hipMemcpy(&match[0], dmatch, sizeof(int)*graph.nrVertices, hipMemcpyDeviceToHost) != hipSuccess)
	{
		cerr << "Unable to retrieve data!" << endl;
		throw exception();
	}
}