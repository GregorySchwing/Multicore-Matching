#include "hip/hip_runtime.h"
/*
Copyright 2022, Gregory Schwing.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "TreeBuilder.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

// Alternative to sorting the full paths.  The full paths are indicated by a value >= 0.
__global__ void PopulateTreeKernel(int nrVertices, 
                                int k, 
                                int *deviceTreeRows, 
                                int *deviceTreeColumns, 
                                int *dforwardlinkedlist, 
                                int *dbackwardlinkedlist, 
                                int *dlength){
const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 3 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int pathIndex = atomicAdd(&deviceTreeRows[0], 1);

    if (pathIndex >= k)
        return;

    int first = threadID;
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];
    int fourth = dforwardlinkedlist[third];

    //printf("threadID %d wrote %d %d %d %d\n", threadID, first, second, third, fourth);

    // Test from root for now, this code can have an arbitrary root though
    deviceTreeColumns[4*pathIndex + 0] = first;
    deviceTreeColumns[4*pathIndex + 1] = second;
    deviceTreeColumns[4*pathIndex + 2] = third; 
    deviceTreeColumns[4*pathIndex + 3] = fourth;       
}


__global__ void DetectAndSetPendantPathsCase3(int nrVertices, 
                                            int k,
                                            int *deviceDynamicRows,
                                            int *deviceDynamicColumns,
                                            int *match, 
                                            int *dforwardlinkedlist, 
                                            int *dbackwardlinkedlist, 
                                            int *dlength){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int dynamicIndex;
    // If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 2 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];
    int third = dforwardlinkedlist[second];
    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 3){
        dynamicIndex = atomicAdd(deviceDynamicRows, 1); 
        // Prevent OOB
        if (dynamicIndex < k){
            deviceDynamicColumns[dynamicIndex] = first;
        } else {
            atomicSub(deviceDynamicRows, 1);
        }
    } else if (match[third] == 3){
        dynamicIndex = atomicAdd(deviceDynamicRows, 1); 
        // Prevent OOB
        if (dynamicIndex < k){
            deviceDynamicColumns[dynamicIndex] = third;
        } else {
            atomicSub(deviceDynamicRows, 1);
        }
    }
}

__global__ void DetectAndSetPendantPathsCase4(int nrVertices, 
                                            int k,
                                            int *deviceDynamicRows,
                                            int *deviceDynamicColumns,
                                            int *match, 
                                            int *dforwardlinkedlist, 
                                            int *dbackwardlinkedlist, 
                                            int *dlength){
	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
    int dynamicIndex;

	// If not a head to a path of length 4, return (leaving the headindex == -1)
    if (threadID >= nrVertices || 
        dlength[threadID] != 1 || 
        dbackwardlinkedlist[threadID] != threadID) 
            return;

    int first = dforwardlinkedlist[threadID];
    int second = dforwardlinkedlist[first];

    // Color == 2 if blue vertex has no unmatched neighbors
    // This avoids iterating over all degrees, but it is possible
    // to miss some vertices which could be pendant but are red not blue.
    if (match[first] == 3){
        dynamicIndex = atomicAdd(deviceDynamicRows, 1); 
        // Prevent OOB
        if (dynamicIndex < k){
            deviceDynamicColumns[dynamicIndex] = first;
        } else {
            atomicSub(deviceDynamicRows, 1);
        }
    } else if (match[second] == 3){
        dynamicIndex = atomicAdd(deviceDynamicRows, 1); 
        // Prevent OOB
        if (dynamicIndex < k){
            deviceDynamicColumns[dynamicIndex] = second;
        } else {
            atomicSub(deviceDynamicRows, 1);
        }
    }
}                       


void TreeBuilder::PopulateTree(int nrVertices, 
                                int threadsPerBlock, 
                                int k, 
                                int *deviceTreeRows, 
                                int *deviceTreeColumns, 
                                int *deviceDynamicRows, 
                                int *deviceDynamicColumns,
                                int *dmatch,
                                int *dforwardlinkedlist, 
                                int *dbackwardlinkedlist, 
                                int *dlength){
    int blocksPerGrid = (nrVertices + threadsPerBlock - 1)/threadsPerBlock;
    PopulateTreeKernel<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                            k, 
                                                            deviceTreeRows, 
                                                            deviceTreeColumns,
                                                            dforwardlinkedlist,
                                                            dbackwardlinkedlist, 
                                                            dlength);

    DetectAndSetPendantPathsCase3<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                        k, 
                                                        deviceDynamicRows, 
                                                        deviceDynamicColumns,
                                                        dmatch,
                                                        dforwardlinkedlist,
                                                        dbackwardlinkedlist, 
                                                        dlength);

    DetectAndSetPendantPathsCase4<<<blocksPerGrid, threadsPerBlock>>>(nrVertices,
                                                        k, 
                                                        deviceDynamicRows, 
                                                        deviceDynamicColumns,
                                                        dmatch,
                                                        dforwardlinkedlist,
                                                        dbackwardlinkedlist, 
                                                        dlength); 
}